#include "hip/hip_runtime.h"
// cudadecoder/cuda-decoder-kernels.cu
//
// Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
// Hugo Braun, Justin Luitjens, Ryan Leary
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hipcub/hipcub.hpp>
#include "cuda-decoder-kernels.h"
#include "cuda-decoder-kernels-utils.h"

namespace kaldi {
namespace cuda_decoder {

// Initialize the hashmap with NO_VAL
// Called in InitDeviceData, when building the CudaDecoder object
__global__ void init_hashmap_kernel(DeviceParams cst_dev_params) {
  const int max_nlanes = cst_dev_params.max_nlanes;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, max_nlanes) {
    const int capacity = cst_dev_params.hashmap_capacity;
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(idx, capacity) {
      cst_dev_params.d_hashmap_values.lane(ilane)[idx] =
          KALDI_CUDA_DECODER_HASHMAP_NO_VAL;
    }
  }
}

// Initialize initial channel on  device
// Called by ComputeInitialChannel
// It is NOT called in InitDecoding
// In InitDecoding we will clone the initial channel into the channel we called
// InitDecoding on
// Here we are actually creating this initial channel
// we do that once in the CudaDecoder constructor.
//
// The initial channel is the state of a channel when
// it will start decoding a new utterance
// thread (1, 1, 1)
// blocks(1, 1, 1);
__global__ void initialize_initial_lane_kernel(DeviceParams cst_dev_params) {
  const int init_ichannel = cst_dev_params.init_channel_id;
  const int init_ilane = 0;
  ChannelCounters *init_channel_counters =
      cst_dev_params.d_channels_counters.channel(init_ichannel);
  LaneCounters *lane_counters =
      cst_dev_params.d_lanes_counters.lane(init_ilane);

  // Making the data look like an ExpandArcsEmitting just executed,
  // and put the StartState in the aux_q. We will then pick up a normal
  // execution from there
  // (calling PruneAndPreprocess, then ExpandArcsNonEmitting..)
  lane_counters->aux_q_end = 0;
  lane_counters->aux_q_requested = 0;
  lane_counters->post_expand_aux_q_end = 1;
  lane_counters->main_q_global_offset = 0;
  lane_counters->main_q_local_offset = 0;
  lane_counters->main_q_n_extra_prev_tokens = 0;
  lane_counters->int_cutoff = INT_MAX;
  lane_counters->main_q_n_emitting_tokens = 0;  // all non emitting
  lane_counters->int_beam = floatToOrderedInt(cst_dev_params.default_beam);
  lane_counters->main_q_narcs_and_end = {0, 0};
  lane_counters->main_q_requested = 0;
  lane_counters->prev_arg_min_int_cost = 0;
  const StateId init_state = cst_dev_params.init_state;
  const CostType init_cost = cst_dev_params.init_cost;
  IntegerCostType int_init_cost = floatToOrderedInt(init_cost);
  cst_dev_params.d_aux_q_state_and_cost.lane(init_ilane)[0] = {init_state,
                                                               int_init_cost};
  lane_counters->min_int_cost = int_init_cost;
  CostType cutoff = orderedIntToFloat(int_init_cost);
  lane_counters->int_cutoff =
      floatToOrderedInt(cutoff + cst_dev_params.default_beam);
  cst_dev_params.d_aux_q_info.lane(init_ilane)[0] = {INT_MIN, -1};
}

// Called by InitDecoding
// Called when some channels will start decoding a new utterance
// do everything that's needed to do on the device to start decoding a new
// utterance with those channels
// It clones the initial channel (created in initialize_initial_lane_kernel)
// into the channels we want to InitDecoding on
__global__ void init_decoding_on_device_kernel(DeviceParams cst_dev_params,
                                               KernelParams params) {
  const int init_ichannel = cst_dev_params.init_channel_id;

  const ChannelCounters *init_channel_counters =
      cst_dev_params.d_channels_counters.channel(init_ichannel);
  const int32 init_main_q_end =
      init_channel_counters->prev_main_q_narcs_and_end.y;
  const int32 nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(idx, init_main_q_end) {
      const LaneCounters *lane_counters =
          cst_dev_params.d_lanes_counters.lane(ilane);
      const int32 ichannel = lane_counters->channel_to_compute;
      cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[idx] =
          cst_dev_params.d_main_q_state_and_cost.channel(init_ichannel)[idx];
      cst_dev_params.d_main_q_degrees_prefix_sum.channel(ichannel)[idx] =
          cst_dev_params.d_main_q_degrees_prefix_sum.channel(
              init_ichannel)[idx];
      cst_dev_params.d_main_q_arc_offsets.channel(ichannel)[idx] =
          cst_dev_params.d_main_q_arc_offsets.channel(init_ichannel)[idx];
      if (idx == 0) {
        ChannelCounters *channel_counters =
            cst_dev_params.d_channels_counters.channel(ichannel);
        channel_counters->prev_main_q_narcs_and_end =
            init_channel_counters->prev_main_q_narcs_and_end;
        channel_counters->prev_main_q_n_extra_prev_tokens =
            init_channel_counters->prev_main_q_n_extra_prev_tokens;
        channel_counters->prev_main_q_global_offset = 0;
        channel_counters->prev_main_q_extra_prev_tokens_global_offset = 0;
        channel_counters->prev_beam = cst_dev_params.default_beam;
      }
    }
  }
}

// Context switch : load
// Called by LoadChannelsStateToLanes
// THREADS : (1, 1, 1)
// BLOCKS : (1, nlanes_used, 1)
__global__ void load_channels_state_in_lanes_kernel(DeviceParams cst_dev_params,
                                                    KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    const ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);
    int2 main_q_narcs_and_end = channel_counters->prev_main_q_narcs_and_end;
    lane_counters->main_q_narcs_and_end = main_q_narcs_and_end;
    lane_counters->main_q_n_extra_prev_tokens =
        channel_counters->prev_main_q_n_extra_prev_tokens;
    CostType beam = channel_counters->prev_beam;
    IntegerCostType int_beam = floatToOrderedInt(beam);
    lane_counters->int_beam = int_beam;
    lane_counters->adaptive_int_beam_with_validity_index.x = int_beam;
    lane_counters->adaptive_int_beam_with_validity_index.y =
        cst_dev_params.adaptive_beam_static_segment;
    lane_counters->main_q_global_offset =
        channel_counters
            ->prev_main_q_global_offset;  // we'll update it after emitting
    lane_counters->main_q_extra_prev_tokens_global_offset =
        channel_counters->prev_main_q_extra_prev_tokens_global_offset;
  }
}

// Context switch : store
// Called by SaveChannelsStateFromLanes
// THREADS : (1, 1, 1)
// BLOCKS : (1, nchannel_to_compute, 1)
__global__ void save_channels_state_from_lanes_kernel(
    DeviceParams cst_dev_params, KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    const LaneCounters *lane_counters =
        cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);
    channel_counters->prev_main_q_global_offset =
        lane_counters->main_q_global_offset;
    channel_counters->prev_main_q_extra_prev_tokens_global_offset =
        lane_counters->main_q_extra_prev_tokens_global_offset;
    channel_counters->prev_main_q_narcs_and_end =
        lane_counters->main_q_narcs_and_end;
    channel_counters->prev_main_q_n_extra_prev_tokens =
        lane_counters->main_q_n_extra_prev_tokens;
    channel_counters->prev_beam = orderedIntToFloat(lane_counters->int_beam);
  }
}

// compute_lane_offsets_kernel
// the kernel concatenate_lanes_data concatenates multiple array into a single
// continuous array
// compute_lane_offsets_kernel computes the offset of each array into this
// continous array
// This kernel is 1D : the lanes are on the X dimension, because we want to
// compute the offset of those lanes
__global__ void compute_lane_offsets_kernel(DeviceParams cst_dev_params,
                                            KernelParams params) {
  typedef hipcub::BlockScan<int4, KALDI_CUDA_DECODER_1D_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;

  const int nlanes = params.nlanes_used;
  int4 sum_so_far = {0, 0, 0, 0};
  KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(
      block_offset, thread_idx,
      nlanes + 1) {  // +1 because we are doing an exclusive sum, and we want
                     // all the values
    int32 ilane = block_offset + thread_idx;
    int4 zero4 = {0, 0, 0, 0};
    int4 lane_offsets = zero4;
    if (ilane < nlanes) {  // nlanes, not nlanes+1, because we cannot read +1
                           // values (undefined)
      LaneCounters *d_lane_counters =
          cst_dev_params.d_lanes_counters.lane(ilane);
      int32 main_q_end = d_lane_counters->main_q_narcs_and_end.y;
      int32 n_emitting_tokens = d_lane_counters->main_q_n_emitting_tokens;
      int32 main_q_n_extra_prev_tokens =
          d_lane_counters->main_q_n_extra_prev_tokens;
      lane_offsets = {main_q_end, n_emitting_tokens, main_q_n_extra_prev_tokens,
                      0};
    }
    int4 block_aggregate;
    BlockScan(temp_storage)
        .ExclusiveScan(lane_offsets, lane_offsets, zero4, PlusPlusPlusPlus(),
                       block_aggregate);
    PlusPlusPlusPlus pppp;
    lane_offsets = pppp(lane_offsets, sum_so_far);
    sum_so_far = pppp(sum_so_far, block_aggregate);
    if (ilane < (nlanes + 1)) {  // nlanes+1, to write the output
      LaneCounters *d_lane_counters =
          cst_dev_params.d_lanes_counters.lane(ilane);
      LaneCounters *h_lane_counters =
          cst_dev_params.h_lanes_counters.lane(ilane);
      h_lane_counters->main_q_end_lane_offset =
          d_lane_counters->main_q_end_lane_offset = lane_offsets.x;
      h_lane_counters->main_q_n_emitting_tokens_lane_offset =
          d_lane_counters->main_q_n_emitting_tokens_lane_offset =
              lane_offsets.y;
      h_lane_counters->main_q_n_extra_prev_tokens_lane_offset =
          d_lane_counters->main_q_n_extra_prev_tokens_lane_offset =
              lane_offsets.z;
    }
    __syncthreads();  // reusing temp_storage
  }
}

// concatenate_lanes_data
// Called by PerformConcatenatedCopy
// Creates a concatenate array into concat,
// by concatenating all the arrays src.lane(ilane)
// for ilane=0..params.nlanes_used
// Used to prepare data for copy to Host. We want to avoid small Device2Host
// copies.
template <typename T>
__global__ void concatenate_lanes_data_kernel(DeviceParams cst_dev_params,
                                              KernelParams params,
                                              LaneMatrixView<T> src, T *concat,
                                              int32 *lane_offsets) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    const int32 stride =
        sizeof(LaneCounters) / sizeof(int32);  // offsets are in LaneCounters
    int32 beg = *(lane_offsets + ilane * stride);
    int32 end = *(lane_offsets + (ilane + 1) * stride);
    int32 vec_size = end - beg;
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(idx, vec_size) {
      T d = src.lane(ilane)[idx];
      concat[beg + idx] = d;
    }
  }
}

// nonemitting_preprocess_and_contract_kernel
// Called from PruneAndPreprocess
// This kernels prune the aux_q, move the survival tokens to the main_q,
// and add the preprocessing information necessary for the next ExpandArcs
// (the expand that follows PruneAndPreprocess is always non-emitting)
// It prunes the tokens using the cutoff, and prepare the data necessary for
// ExpandArcs:
// d_main_q_degrees_prefix_sum, d_main_q_arc_offsets_
// The prefix sum is done in one-pass here, using a trick (we compute the prefix
// sum
// as we fill the main_q)
__global__ void nonemitting_preprocess_and_contract_kernel(
    DeviceParams cst_dev_params, KernelParams params) {
  typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_1D_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage sh_temp_storage;
  // We need to move the survival tokens to the main_q
  //
  // sh_main_q_global_block_offset has two purposes :
  // (1) to know where to store the survival tokens in the main_q
  // (2) to perform the prefix sum degrees (of the survival tokens)
  __shared__ int2 sh_main_q_global_block_offset;
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 aux_q_end = lane_counters->post_expand_aux_q_end;
    const IntegerCostType int_cutoff = lane_counters->int_cutoff;
    // Keeping whole CTA alive. We'll use __syncthreads()
    KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx,
                                                   aux_q_end) {
      const int32 aux_q_idx = block_offset + thread_idx;
      const int32 ichannel = lane_counters->channel_to_compute;
      int32 degree = 0;
      int32 arc_start = -1;
      StateId token_state;
      IntegerCostType token_int_cost;
      // We've kept the whole CTA alive. Now we keep only those will a valid
      // token
      if (aux_q_idx < aux_q_end) {
        const int2 both =
            cst_dev_params.d_aux_q_state_and_cost.lane(ilane)[aux_q_idx];
        token_state = both.x;
        token_int_cost = both.y;

        if (token_int_cost < int_cutoff) {
          // We'll keep that token. Loading its arc degree/csr offset now.
          arc_start = cst_dev_params.d_arc_ne_offsets[token_state];
          const int32 arc_end =
              cst_dev_params.d_arc_ne_offsets[token_state + 1];
          degree = arc_end - arc_start;
        }
      }

      // If we've set a different arc_start,
      // this thread has a valid unpruned token
      int32 is_pruned = (arc_start == -1);

      // We now know which tokens will be moved to the main_q, the remaining
      // will be pruned
      // we now compute a prefix sum inside the CUDA block to determine the
      // local indexes of the unpruned tokens
      // the first unpruned token will have a index of 0, the second 1, ...
      // We also need to compute the prefix sum of the arc degrees
      // we start by doing a local prefix sum inside the CUDA block
      int2 block_prefix_sum_narcs_and_end = {degree, (is_pruned ? 0 : 1)};
      const int2 zero2 = {0, 0};

      // Computing the prefix sum (exclusive)
      BlockScan(sh_temp_storage)
          .ExclusiveScan(block_prefix_sum_narcs_and_end,
                         block_prefix_sum_narcs_and_end, zero2, PlusPlus());

      if (KALDI_CUDA_DECODER_IS_LAST_1D_THREAD()) {
        // This conditional branch is entered by the last thread
        // Because it is the last, the prefix_sum of that thread contains the
        // sum of all elements

        // We also add the value from this thread - the prefix sum is exclusive
        // For the sum, we want it inclusive
        int2 block_sum = block_prefix_sum_narcs_and_end;
        block_sum.x += degree;
        block_sum.y += is_pruned ? 0 : 1;

        // Doing two things at the same time :
        // requesting a spot in the main_q to store the survival tokens from
        // this CTA
        // We also increment the narcs value. atomic64.x will contain the number
        // of
        // arcs in the main_q up until the atomic64.y index
        // That's all we need to finish our prefix sum. We add this global
        // offset.

        // First atomic to check if we are not overflowing main_q.
        int block_offset =
            atomicAdd(&lane_counters->main_q_requested, block_sum.y);

        // Verify that we do not overflow
        if (block_offset + block_sum.y < cst_dev_params.main_q_capacity) {
          // we don't overflow we can safely grab a spot in the main_q
          sh_main_q_global_block_offset =
              atomicAddI2(&lane_counters->main_q_narcs_and_end, block_sum);
        } else {
          // our update would overflow
          lane_counters->q_overflow |= OVERFLOW_MAIN_Q;  // for the host
          sh_main_q_global_block_offset.y =
              cst_dev_params.main_q_capacity;  // used as flag to broadcast the
                                               // information in the CTA
        }
      }

      // Syncing because :
      // - Broadcasting sh_main_q_global_block_offset
      // - We may reuse sh_temp_storage (cf CUB doc)
      __syncthreads();

      // Checking if we are overflowing the main_q
      // All threads are executing the next line
      if (sh_main_q_global_block_offset.y == cst_dev_params.main_q_capacity)
        goto end_lane;  // done for this lane

      // If we are executing the following lines it means that we are not
      // overflowing the queue
      // We then continue what we were doing
      if (!is_pruned) {
        bool moving_emitting_tokens = (lane_counters->main_q_local_offset == 0);
        // we will move our unpruned token to the main_q, at index main_q_idx
        InfoToken tok_info = cst_dev_params.d_aux_q_info.lane(ilane)[aux_q_idx];
        const int32 main_q_idx =
            sh_main_q_global_block_offset.y + block_prefix_sum_narcs_and_end.y;
        CostType acoustic_cost = 0.0f;
        if (moving_emitting_tokens && tok_info.arc_idx != -1) {
          const int32 arc_ilabel =
              cst_dev_params.d_arc_pdf_ilabels[tok_info.arc_idx];
          acoustic_cost = -lane_counters->loglikelihoods[arc_ilabel];
        }
        cst_dev_params.d_main_q_info.lane(ilane)[main_q_idx] = tok_info;

        // Moving the token to the main q
        cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[main_q_idx] = {
            token_state, token_int_cost};
        cst_dev_params.d_main_q_acoustic_cost.lane(ilane)[main_q_idx] =
            acoustic_cost;
        // Saving the global prefix sum
        const int32 prefix_sum_narcs =
            sh_main_q_global_block_offset.x + block_prefix_sum_narcs_and_end.x;
        cst_dev_params.d_main_q_degrees_prefix_sum.channel(
            ichannel)[main_q_idx] = prefix_sum_narcs;
        // Saving the CSR arc offset for that token's state
        // it will be used by the expand kernel, and avoid doing a new random
        // memory access in the expand kernel
        cst_dev_params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx] =
            arc_start;
      }
    }

  end_lane:;  // empty statement
  }
}

// GetAdaptiveBeam is used in ExpandArcs
// When we generate new tokens by traversing arcs, 
// we can end up creating a lot of tokens, if the current frame 
// generated loglikelihoods too uniform for instance (we don't have
// any good tokens that will reduce the cutoff, so we end up generating
// a lot of tokens)
// To avoid overflowing the aux_q, we apply a decreasing beam.
// With aux_q_end being the current aux_q size, we have a decrease function f, with
// adaptive_beam = f(aux_q_end)
// f is a decreasing piecewise constant function
// Please note that when processing tokens, we usually have dozens of thousands of threads
// generating tokens. Those are already in flight, and will not reload the beam immediatly.
// It means that we need to start reducing the beam as soon as we detect that we are generating more tokens than
// expected. 
// We can configure the function f using KALDI_CUDA_DECODER_ADAPTIVE_BEAM_STATIC_SEGMENT
// and KALDI_CUDA_DECODER_ADAPTIVE_BEAM_NSTEPS.
// We will use default_beam for the first max_tokens_per_frame/KALDI_CUDA_DECODER_ADAPTIVE_BEAM_STATIC_SEGMENT
// tokens in the aux_q.
// Once we reach that number, we will decrease the adaptive beam linearly from default_beam to 0,
// using KALDI_CUDA_DECODER_ADAPTIVE_BEAM_NSTEPS steps
//
// x-axis : aux_q_end. How much tokens are already in the aux_q
// y-axis : adaptive_beam = f(aux_q_end)
// default_beam _| ________________
//               |               /\ _________
//               |                |          _________
//            0 _|   static_segment                   _________
//               |________________________________________________
//               |                                             |     
//   aux_q_end=  0                                    max_tokens_per_frame
// We have :     
// static_segment = max_tokens_per_frame/KALDI_CUDA_DECODER_ADAPTIVE_BEAM_STATIC_SEGMENT
// and KALDI_CUDA_DECODER_ADAPTIVE_BEAM_NSTEPS = 3
__device__ void UpdateAdaptiveBeam(const DeviceParams &cst_dev_params,
                                   const int aux_q_index_block_offset,
                                   IntegerCostType min_int_cost,
                                   int2 *adaptive_int_beam_with_validity_index,
                                   LaneCounters *lane_counters) {
  int32 beam_valid_until_idx = adaptive_int_beam_with_validity_index->y;
  if (aux_q_index_block_offset < beam_valid_until_idx) return;  // nothing to do

  CostType beam = orderedIntToFloat(adaptive_int_beam_with_validity_index->x);
  while (aux_q_index_block_offset >= beam_valid_until_idx) {
    beam /= 2;
    beam_valid_until_idx += cst_dev_params.adaptive_beam_bin_width;
  }

  IntegerCostType new_int_cutoff = (min_int_cost < INT_MAX)
      ? floatToOrderedInt(orderedIntToFloat(min_int_cost) + beam)
      : INT_MAX;
  IntegerCostType int_beam = floatToOrderedInt(beam);
  adaptive_int_beam_with_validity_index->x = int_beam;
  adaptive_int_beam_with_validity_index->y = beam_valid_until_idx;
  // We can have races between the two atomics
  // However the worst than can happen is a CTA might delay updating the beam
  // This is not a critical bug. However, once we have a floatToOrderedInt
  // that is generating unsigned ints, we could merge the two atomics into a
  // single atomic64
  atomicMin(&lane_counters->adaptive_int_beam_with_validity_index.x, int_beam);
  atomicMax(&lane_counters->adaptive_int_beam_with_validity_index.y,
            beam_valid_until_idx);
  atomicMin(&lane_counters->int_cutoff, new_int_cutoff);
}

// One CTA / lane
__global__ void reset_for_frame_and_estimate_cutoff_kernel(
    DeviceParams cst_dev_params, KernelParams params) {
  typedef hipcub::BlockReduce<CostType, KALDI_CUDA_DECODER_1D_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);
    if (threadIdx.x == 0) {
      const CostType current_beam = orderedIntToFloat(lane_counters->int_beam);
      // Do some initialization
      lane_counters->q_overflow = OVERFLOW_NONE;
      lane_counters->main_q_n_emitting_tokens = INT_MAX;
      lane_counters->int_cutoff = INT_MAX;
      lane_counters->min_int_cost = INT_MAX;
      lane_counters->q_overflow = OVERFLOW_NONE;
      lane_counters->int_relative_cost = INT_MAX;
      lane_counters->aux_q_requested = 0;
      lane_counters->main_q_requested = 0;
      lane_counters->main_q_local_offset = 0;
      lane_counters->compute_max_active =
          false;  // will be set to true if necessary
      channel_counters->min_int_cost_and_arg_with_final.x =
          INT_MAX;  // it will be set with atomicMins
      const CostType new_beam =
          fmin(cst_dev_params.default_beam,
               current_beam * KALDI_CUDA_DECODER_ADAPTIVE_BEAM_RECOVER_RATE);
      lane_counters->int_beam = floatToOrderedInt(new_beam);
    }
    const int32 prev_arg_min = lane_counters->prev_arg_min_int_cost;
    int2 both =
        cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[prev_arg_min];
    int32 int_cost = both.y;
    CostType previous_cost = orderedIntToFloat(int_cost);
    const int32 prev_arg_min_state = both.x;
    int32 arc_start = cst_dev_params.d_arc_e_offsets[prev_arg_min_state];
    int32 arc_end = cst_dev_params.d_arc_e_offsets[prev_arg_min_state + 1];
    int32 narcs = arc_end - arc_start;
    // no loop - we only process the first KALDI_CUDA_DECODER_1D_BLOCK arcs
    // we just want an estimate
    CostType total_cost = FLT_MAX;
    if (threadIdx.x < narcs) {
      int32 iarc = arc_start + threadIdx.x;
      CostType arc_fixed_cost = cst_dev_params.d_arc_weights[iarc];
      const int32 arc_ilabel = cst_dev_params.d_arc_pdf_ilabels[iarc];
      CostType acoustic_cost = -lane_counters->loglikelihoods[arc_ilabel];
      total_cost = previous_cost + arc_fixed_cost +
                   acoustic_cost;  // +0.0f, best prev cost is normalized to 0
    }

    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(bin_id, KALDI_CUDA_DECODER_HISTO_NBINS) { 
      cst_dev_params.d_histograms.lane(ilane)[bin_id] = 0; // reset for this frame
    }

    CostType min = BlockReduce(temp_storage).Reduce(total_cost, hipcub::Min());
    if (narcs > 0 && threadIdx.x == 0) {
      // narcs > 0 to have at least one valid element in the reduce
      CostType new_cutoff = min + orderedIntToFloat(lane_counters->int_beam);
      IntegerCostType new_int_cutoff = floatToOrderedInt(new_cutoff);
      lane_counters->int_cutoff = new_int_cutoff;
      lane_counters->min_int_cost = floatToOrderedInt(min);
    }
  }
}
// ExpandArc kernel
// This kernel does the actual work of traversing arcs
//
// Pseudo code :
// for all token tok in main_q[main_q_offset...end]:
//      u = tok.next_state
//      for all arc a(u->v) in the FST:
//          v_cost = tok.cost + a.cost + accoustic_cost
//
//          if v_cost < cutoff and v_cost < best_state_cost[v]
//              generate token associated to v, add to aux_q
//              if necessary update cutoff
//              if aux_q is getting full, reduce beam
//
// For more information please refer to http://kaldi-asr.org/doc/decoders.html
//
// ExpandArc rely on some preprocessed data to be able to function
// for instance, it needs the prefix sum of the arc degree of all token.state in
// the main_q
// We need to call a Preprocess kernel before ExpandArc
//
// ExpandArc is used for both emitting and nonemitting phases
// Differences between emitting and nonemitting :
//      1) params.d_q_arc_offset contains offsets to either emitting or
//      nonemitting arcs.
//         It is transparent for this kernel. The differentiation was done in
//         the Preprocess kernel,
//         which is responsible for filling the params.d_q_arc_offset array
//      2) Computation of the acoustic cost. If nonemitting, it is equal to 0.
//      If emitting, we need
//         to use values from the acoustic model (through the d_loglikelihoods
//         array)
//
// Note : ExpandArc is not the only kernel able to traverse arcs.
// FinalizeProcessNonemitting contains a simplified version of expand for only
// one CUDA block
template <bool IS_EMITTING>
__global__ void expand_arcs_kernel(DeviceParams cst_dev_params,
                                   KernelParams params) {
  // BlockScan that we will use to compute token indexes in the output queue,
  // and to find the min cost in the block
  typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_1D_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage sh_temp_storage_scan;

  // This kernel writes the new token to the output queue aux_q
  // We will request a spot to store all the new tokens created by threads in
  // this CUDA block
  // sh_aux_q_index_block_offset indicates where to store them in the aux_q
  // tokens created in this CUDA block will be store in :
  // aux_q[sh_aux_q_index_block_offset], aux_q[sh_aux_q_index_block_offset + 1],
  __shared__ int32 sh_aux_q_index_block_offset;
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 main_q_offset = lane_counters->main_q_local_offset;
    const int32 main_q_end = lane_counters->main_q_narcs_and_end.y;
    const int32 total_narcs = lane_counters->main_q_narcs_and_end.x;
    KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx,
                                                   total_narcs) {
      int2 adaptive_int_beam_with_validity_index =
          lane_counters->adaptive_int_beam_with_validity_index;
      const int32 ichannel = lane_counters->channel_to_compute;
      // Important : this thread is not responsible for a token in the input
      // queue main_q
      // but for an arc, going out of a token in the main_q
      // The main_q contains in total total_narcs
      // and this thread will compute the main_q_arc_index-th arc of the main_q
      // For instance, first thread in the grid with threadIdx.x == 0 and
      // blockIdx.x == 0
      // will process the first arc of the token in main_q[main_q_offset + 0]
      // (if that token has at least one arc)
      //
      // This insure a perfect one thread = one arc load balancing
      // but we have work to do to know exactly which arc is the
      // main_q_arc_index-th arc
      // (what's its source ? its destination ? its arc_idx the FST CSR ?)
      int32 main_q_arc_index = block_offset + thread_idx;
      // We'll need those variables later in the kernel
      // we declare them outside of the "valid_input" scope
      // to be able to access them later
      int32 main_q_idx;
      int32 arc_idx;
      StateId arc_next_state;
      IntegerCostType int_total_cost = INT_MAX;
      if (main_q_arc_index < total_narcs) {
        // Current thread must take care of main_q_arc_index-th arc
        // we need to now what's the source of that arc
        // ie which token.state in main_q does it start from ?
        // We use a binary search in the prefix sum of the token's degree to get
        // that information
        //
        // Example : main_q contains 3 tokens
        // - First token is associated to a state which has 3 outgoing arc
        // - Second token is associated to a state which has 0 outgoing arc
        // - Third token is associated to a state which has 2 outgoing arc
        //
        // We store the degrees in an array :
        // [3, 0, 2]
        //
        // We then compute the exclusive prefix sum of that array :
        // [0, 3, 3, 5]
        //
        // In total, we have 5 arcs in the main_q. ExpandArc will use 5 threads.
        //
        // Let's say we are the fifth thread in ExpandArc.
        // we have threadIdx.x == 4, and blockIdx.x == 0
        // it gives us main_q_arc_index == 4
        // From there we have no idea what we're supposed to do next, we need to
        // have information about the
        // arc that we're supposed to traverse
        //
        // To do that, we look for the maximum index maxle_i in the prefix sum
        // array such prefix_sum[i] <= 4
        //
        // [0, 3, 3, 5]
        //          |
        //         here
        // maxle_i = 2
        // it means that our source token is at index 2 in the main_q
        // and we are computing the arc at index (main_q_arc_index -
        // prefix_sum[maxle_i]) of that token
        // ie the arc at index (4-3) = 1, the second arc of the second token in
        // main_q

        // Searching for the source of the arc that we will process
        // (main_q_arc_index)
        // we could preprocess the search in the preprocess kernels - for now
        // this kernel is fast enough
        const int32 *degrees_prefix_sum =
            cst_dev_params.d_main_q_degrees_prefix_sum.channel(ichannel);
        main_q_idx = binsearch_maxle(degrees_prefix_sum, main_q_arc_index,
                                     main_q_offset, main_q_end - 1);

        // state_first_arc_idx_in_main_q
        // d_main_q_degrees_prefix_sum contains the prefix sum of the
        // degrees of all tokens in the main_q
        // d_main_q_degrees_prefix_sum[main_q_idx] contains the number of arc
        // in the main_q until that token
        const int32 state_first_arc_idx_in_main_q =
            degrees_prefix_sum[main_q_idx];

        // arc_offset_start is the offset in the CSR, to find the arcs
        // related to the state main_q_state_[main_q_idx]
        // it was set by the preprocess kernel
        const int32 arc_offset_start =
            cst_dev_params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx];

        // local_arc_index is the arc index for that state
        // if local_arc_index == 2, we will process the second arc
        // of state main_q_state_[main_q_idx]
        const int32 local_arc_index =
            main_q_arc_index - state_first_arc_idx_in_main_q;

        // corresponding arc_idx in the FST
        arc_idx = arc_offset_start + local_arc_index;

        // Destination of that arc
        arc_next_state = cst_dev_params.d_arc_nextstates[arc_idx];

        // Building the total cost incrementally
        // we'll add the acoustic cost and the old token's cost
        const CostType arc_fixed_cost = cst_dev_params.d_arc_weights[arc_idx];
        const CostType prev_token_cost = orderedIntToFloat(
            cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[main_q_idx]
                .y);
        CostType total_cost = prev_token_cost + arc_fixed_cost;
        const int32 prev_state =
            cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[main_q_idx]
                .x;
        if (IS_EMITTING) {
          const int32 arc_ilabel = cst_dev_params.d_arc_pdf_ilabels[arc_idx];
          CostType acoustic_cost = -lane_counters->loglikelihoods[arc_ilabel];
          total_cost += acoustic_cost;
        }
        int_total_cost = floatToOrderedInt(total_cost);

        // If the total_cost is too large compared to our cutoff (beam search)
        // then let's drop it
        const IntegerCostType int_cutoff = lane_counters->int_cutoff;
        if (int_total_cost >= int_cutoff) int_total_cost = INT_MAX;
      }

      // If int_total_cost < INT_MAX, it means that :
      // - this thread had a valid input (main_q_arc_index < total_narcs)
      // - the total_cost of the generated token is < cutoff
      // We will then add that new token in the output queue, aux_q
      // We need to know where to put that token in the aux_q
      // we'll first compute its index inside the CUDA block
      // the first valid output token in the CUDA block will have index 0,
      // the second index 1... We compute that using a prefix sum
      //
      // We also need to find the overall min cost in the CUDA block
      // a prefix sum is a scan operation, and a min a reduce operation
      // we can perform a reduce operation using a scan (using the last value)
      // we compute the prefix sum and the min in one scan, using the data
      // struct CostTypeAndInt
      const int32 has_successor = (int_total_cost < INT_MAX) ? 1 : 0;

      int2 int_cost_and_index = {int_total_cost, has_successor};
      BlockScan(sh_temp_storage_scan)
          .InclusiveScan(int_cost_and_index, int_cost_and_index, MinPlus());
      if (KALDI_CUDA_DECODER_IS_LAST_1D_THREAD()) {
        // We are in a divergent branch
        // This is the last thread. The last value of the inclusive scan is the
        // total
        const int32 total_successors_in_block = int_cost_and_index.y;
        // Requesting a spot of size total_successors_in_block in the aux_q

        // note:  using 2 atomics here to avoid adding another kernel
        // first request more space
        const int aux_q_index_block_offset = atomicAdd(
            &lane_counters->aux_q_requested, total_successors_in_block);

        // check for overflow in aux_q
        // We try to prevent an overflow from happening using an adaptive beam
        // (cf GetAdaptiveBeam)
        if (aux_q_index_block_offset + total_successors_in_block <
            cst_dev_params.aux_q_capacity) {
          // no overflow

          // grab the aux_q offset
          sh_aux_q_index_block_offset =
              atomicAdd(&lane_counters->aux_q_end, total_successors_in_block);

          // We are not overflowing the queue, updating the global values
            IntegerCostType global_min_int_cost = lane_counters->min_int_cost;
            IntegerCostType local_min_int_cost = int_cost_and_index.x;
            // if we found a lower min_cost, update the global value
            if (local_min_int_cost < global_min_int_cost) {
              global_min_int_cost = local_min_int_cost;
              atomicMin(&lane_counters->min_int_cost, global_min_int_cost);
              CostType beam =
                  orderedIntToFloat(adaptive_int_beam_with_validity_index.x);
              IntegerCostType new_int_cutoff = floatToOrderedInt(
                  orderedIntToFloat(local_min_int_cost) + beam);
              atomicMin(&lane_counters->int_cutoff, new_int_cutoff);
            }
            int32 beam_valid_until_idx =
                adaptive_int_beam_with_validity_index.y;
            if (aux_q_index_block_offset >= beam_valid_until_idx) {
              // This beam is no longer valid. Updating it
              UpdateAdaptiveBeam(
                  cst_dev_params, aux_q_index_block_offset, global_min_int_cost,
                  &adaptive_int_beam_with_validity_index, lane_counters);
            }
        } else {
          // sh_aux_q_index_block_offset is in shared memory
          // its value is currently invalid (overflow)
          // we set it to a special value and use it as a flag to broadcast
          // the fact that we have an overflow and that all threads should exit
          sh_aux_q_index_block_offset = cst_dev_params.aux_q_capacity;

          // Setting the flag for the host. It will be used to print a warning
          // to stderr
          lane_counters->q_overflow |= OVERFLOW_AUX_Q;

          // We do not jump to end_lane now, because only
          // the first thread (threadIdx.x == 0) is executing this
          // We wait until the end of the divergent branch
        }
      }

      // Sync'ing for two reasons :
      // - Broadcasting sh_aux_q_index_block_offset
      // - reusing sh_temp_storage (cf CUB's doc)
      __syncthreads();
      // The only case where we can have that condition met,
      // is if we detected an overflow if the previous lines
      if (sh_aux_q_index_block_offset == cst_dev_params.aux_q_capacity)
        goto end_lane;  // done for this lane
      //
      // If we're executing the following lines it means everything
      // is valid and we are not overflowing the aux_q
      //
      int_cost_and_index.y -= has_successor;  // we want the exclusive sum now
      const int32 aux_q_block_index = int_cost_and_index.y;
      const int32 aux_q_index = sh_aux_q_index_block_offset + aux_q_block_index;
      if (has_successor) {
        // We save the new token to the aux_q
        cst_dev_params.d_aux_q_state_and_cost.lane(ilane)[aux_q_index] = {
            arc_next_state, int_total_cost};
        // Index of the parent token
        // the parent is the token used as input (source of arc)
        // that parent is at index main_q_idx in the GPU memory
        // However, the main_q is emptied before processing a new frame
        // we need to add the offset related to the previous frames index
        // we add cst_dev_params.main_q_global_offset
        const int32 prev_token =
            lane_counters->main_q_global_offset + main_q_idx;
        assert(main_q_idx >= 0 && main_q_idx < cst_dev_params.main_q_capacity);
        cst_dev_params.d_aux_q_info.lane(ilane)[aux_q_index] = {prev_token,
                                                                arc_idx};
      }
    }
  end_lane:;  // ";" is an empty statement
  }
}

// post_expand_kernel
// Called after expand_arcs_kernel
// Takes care of what needs to be done after an expand_arcs_kernel
// execution. Mostly resetting the beam (if adaptive beam was triggered,
// the max_active_ kernels will take care of selecting a good beam),
// resetting the number of arcs in the main_q (we've processed them all),
// etc.
// Threads (1,1,1)
// Blocks (1, nlanes_used, 1)
template <bool IS_EMITTING>
__global__ void post_expand_kernel(DeviceParams cst_dev_params,
                                   KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    LaneCounters *h_lane_counters = cst_dev_params.h_lanes_counters.lane(ilane);
    const int prev_main_q_end = lane_counters->main_q_narcs_and_end.y;
    const int prev_n_extra_prev_tokens =
        lane_counters->main_q_n_extra_prev_tokens;
    const int aux_q_end = lane_counters->aux_q_end;
    CostType min_cost = orderedIntToFloat(lane_counters->min_int_cost);
    // The next step is the contracting step from aux_q to main_q
    // It will need the aux_q_end value. But it will also empty the aux_q
    // We're resetting aux_q_end to 0 now, but we're saving its old value
    // in another place
    lane_counters->post_expand_aux_q_end = aux_q_end;
    h_lane_counters->post_expand_aux_q_end = aux_q_end;       // pinned memory
    h_lane_counters->q_overflow = lane_counters->q_overflow;  // pinned memory
    lane_counters->aux_q_end = 0;
    lane_counters->aux_q_requested = 0;
    // We are done processing those arcs
    lane_counters->main_q_narcs_and_end.x = 0;
    // Resetting the adaptive beam
    lane_counters->adaptive_int_beam_with_validity_index.x =
        lane_counters->int_beam;
    lane_counters->adaptive_int_beam_with_validity_index.y =
        cst_dev_params.adaptive_beam_static_segment;
    CostType beam = orderedIntToFloat(lane_counters->int_beam);
    lane_counters->int_cutoff = floatToOrderedInt(min_cost + beam);
    // If the adaptive beam kicked in, we want to reset the beam
    // the max-active process will take care of selecting the right beam
    if (IS_EMITTING) {
      // the main_q contains the tokens from the previous frame
      // after emitting, we won't use them anymore to create new tokens
      // we reset the main_q
      lane_counters->main_q_narcs_and_end = {0, 0};
      lane_counters->main_q_requested = 0;
      // The main_q was flushed - we need to update the global_offset
      lane_counters->main_q_global_offset += prev_main_q_end;
      if (threadIdx.x == 0 && blockIdx.x == 0)
        lane_counters->main_q_extra_prev_tokens_global_offset +=
            prev_n_extra_prev_tokens;
      // Moving local offset. Tokens created by last expand
      // will be pruned, and survivals will be moved at the end
      // of the main q. Those tokens will be placed after local_offset
      lane_counters->main_q_requested = 0;
      CostType min_cost = orderedIntToFloat(lane_counters->min_int_cost);
      lane_counters->min_histo_cost = min_cost;
      lane_counters->max_histo_cost = min_cost + beam;
      lane_counters->histo_bin_width = beam / (KALDI_CUDA_DECODER_HISTO_NBINS-1);
    } else {
      lane_counters->main_q_local_offset = prev_main_q_end;
      // reset requested to end of queue
      lane_counters->main_q_requested = prev_main_q_end;
    }
  }
}

__global__ void post_contract_and_preprocess_kernel(DeviceParams cst_dev_params,
                                                    KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    LaneCounters *h_lane_counters = cst_dev_params.h_lanes_counters.lane(ilane);
    int2 main_q_narcs_and_end = lane_counters->main_q_narcs_and_end;
    h_lane_counters->main_q_narcs_and_end =
        main_q_narcs_and_end;                                 // pinned memory
    h_lane_counters->q_overflow = lane_counters->q_overflow;  // pinned memory
    atomicMin(&lane_counters->main_q_n_emitting_tokens, main_q_narcs_and_end.y);
  }
}

// Meta-kernel (merging preprocess and expand) but only works with 1 CUDA block
// Used to avoid calling multiple main kernels (such as expand_arcs_kernel)
// for the tail of non emitting (lots of iterations with small number of arcs)
//
// Code is greatly simplified because we use only one CTA / lane
//
// Repeat until new queue empty:
// 1) Preprocess
// 2) Expand arcs
//
// The preprocess stage is not done on the first iteration, because it was
// already done by the ProcessAndContract kernel. We always call
// PruneAndPreprocess before calling FinalizeProcessNonemitting
//
// At the end, this kernel finalize the computation for current frame,
// so that it's ready for next ProcessEmitting
//
// This kernel works, but can be greatly simplified now.
__launch_bounds__(KALDI_CUDA_DECODER_LARGEST_1D_BLOCK, 1) __global__
    void finalize_process_non_emitting_kernel(DeviceParams cst_dev_params,
                                              KernelParams params) {
  typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_LARGEST_1D_BLOCK>
      Int2BlockScan;
  typedef hipcub::BlockScan<int, KALDI_CUDA_DECODER_LARGEST_1D_BLOCK> IntBlockScan;
  __shared__ typename IntBlockScan::TempStorage sh_temp_storage_int_scan;
  __shared__ typename Int2BlockScan::TempStorage sh_temp_storage_int2_scan;

  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);

    int2 both = lane_counters->main_q_narcs_and_end;
    int32 main_q_narcs = both.x;
    int32 main_q_end = both.y;
    int32 main_q_local_offset = lane_counters->main_q_local_offset;
    const int32 main_q_global_offset = lane_counters->main_q_global_offset;
    // aux_q is empty when this kernel is called
    int32 aux_q_end = 0;
    IntegerCostType int_cutoff = lane_counters->int_cutoff;
    while (main_q_narcs > 0) {
      // Step 1 : ExpandArcs
      KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(offset, thread_idx,
                                                     main_q_narcs) {
        const int32 main_q_arc_idx = offset + thread_idx;
        // For details on how this code works, please refer to comments in
        // expand_arcs
        IntegerCostType total_int_cost = INT_MAX;
        int32 arc_idx;
        StateId arc_next_state;
        int32 main_q_idx;
        if (main_q_arc_idx < main_q_narcs) {
          main_q_idx = binsearch_maxle(
              cst_dev_params.d_main_q_degrees_prefix_sum.channel(ichannel),
              main_q_arc_idx, main_q_local_offset, main_q_end - 1);

          const int32 state_first_arc_idx_in_main_q =
              cst_dev_params.d_main_q_degrees_prefix_sum.channel(
                  ichannel)[main_q_idx];
          const int32 arc_offset_start =
              cst_dev_params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx];
          arc_idx = arc_offset_start +
                    (main_q_arc_idx - state_first_arc_idx_in_main_q);

          arc_next_state = cst_dev_params.d_arc_nextstates[arc_idx];
          CostType arc_weight = cst_dev_params.d_arc_weights[arc_idx];
          CostType prev_token_cost =
              orderedIntToFloat(cst_dev_params.d_main_q_state_and_cost
                                    .channel(ichannel)[main_q_idx]
                                    .y);
          total_int_cost = floatToOrderedInt(arc_weight + prev_token_cost);
	  if(total_int_cost < lane_counters->min_int_cost)
            atomicMin(&lane_counters->min_int_cost, total_int_cost);
          if (total_int_cost >= int_cutoff) {
            total_int_cost = INT_MAX;  // above cutoff
          }
        }
        const int32 has_successor = (total_int_cost < INT_MAX) ? 1 : 0;

        int32 local_aux_q_idx;
        int32 nsuccessors;
        IntBlockScan(sh_temp_storage_int_scan)
            .ExclusiveSum(has_successor, local_aux_q_idx,
                          nsuccessors);  // aggregate

        // Checking if we are overflowing the aux_q
        if ((aux_q_end + nsuccessors) >= cst_dev_params.aux_q_capacity) {
          lane_counters->q_overflow |= OVERFLOW_AUX_Q;
          // nothing to revert in global memory
          goto finalize_lane;
        }

        if (has_successor) {
          const int32 aux_q_idx = aux_q_end + local_aux_q_idx;
          const int32 prev_token_idx = main_q_global_offset + main_q_idx;
          cst_dev_params.d_aux_q_state_and_cost.lane(ilane)[aux_q_idx] = {
              arc_next_state, total_int_cost};
          cst_dev_params.d_aux_q_info.lane(ilane)[aux_q_idx] = {prev_token_idx,
                                                                arc_idx};
        }
        aux_q_end += nsuccessors;
        // sync: reusing sh_temp_storage_scan_int
        __syncthreads();
      }

      // Step 2 : PreprocessAndContract
      // Reset for new iteration
      main_q_narcs = 0;
      main_q_local_offset = main_q_end;
      KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(offset, thread_idx,
                                                     aux_q_end) {
        const int32 aux_q_idx = offset + thread_idx;
        int32 degree = 0;
        int32 start = -1;
        StateId token_state;
        IntegerCostType token_int_cost;
        if (aux_q_idx < aux_q_end) {
          int2 both =
              cst_dev_params.d_aux_q_state_and_cost.lane(ilane)[aux_q_idx];
          token_state = both.x;
          token_int_cost = both.y;
          // beam may have changed since generation
          // We are non-emitting in this kernel, using ne offsets
          start = cst_dev_params.d_arc_ne_offsets[token_state];
          int32 end = cst_dev_params.d_arc_ne_offsets[token_state + 1];
          degree = end - start;
        }
        int has_valid_nonpruned_token = (start != -1) ? 1 : 0;
        int2 narcs_and_ntokens_prefix_sum = {degree, has_valid_nonpruned_token};
        int2 aggregate, zero2 = {0, 0};
        Int2BlockScan(sh_temp_storage_int2_scan)
            .ExclusiveScan(narcs_and_ntokens_prefix_sum,
                           narcs_and_ntokens_prefix_sum, zero2, PlusPlus(),
                           aggregate);
        // Checking if we are not overflowing the main_q
        const int32 total_ntokens = aggregate.y;
        if ((main_q_end + total_ntokens) >= cst_dev_params.main_q_capacity) {
          lane_counters->q_overflow |= OVERFLOW_MAIN_Q;
          goto finalize_lane;
        }
        const int32 degree_prefix_sum =
            main_q_narcs + narcs_and_ntokens_prefix_sum.x;
        const int32 degree_sum = aggregate.x;
        main_q_narcs += degree_sum;
        if (has_valid_nonpruned_token) {
          const int32 local_main_q_idx = narcs_and_ntokens_prefix_sum.y;
          const int32 main_q_idx = main_q_end + local_main_q_idx;

          cst_dev_params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx] =
              start;
          cst_dev_params.d_main_q_degrees_prefix_sum.channel(
              ichannel)[main_q_idx] = degree_prefix_sum;
          cst_dev_params.d_main_q_state_and_cost.channel(
              ichannel)[main_q_idx] = {token_state, token_int_cost};
          cst_dev_params.d_main_q_info.lane(ilane)[main_q_idx] =
              cst_dev_params.d_aux_q_info.lane(ilane)[aux_q_idx];
          cst_dev_params.d_main_q_acoustic_cost.lane(ilane)[main_q_idx] =
              0.0f;  // we are always nonemitting in this kernel
        }
        main_q_end += total_ntokens;
        __syncthreads();
      }
      aux_q_end = 0;  // aux_q is now empty
    }

  finalize_lane:
    if (threadIdx.x == 0) {
      // This main_q is now final for that frame
      lane_counters->main_q_narcs_and_end = {0, main_q_end};
      cst_dev_params.h_lanes_counters.lane(ilane)->main_q_narcs_and_end = {
          0, main_q_end};  // pinned memory
    }
  }
}

// GetBestCost :
// Finds all tokens with a cost in [min_cost;min_cost+lattice_beam[
// Add the final_costs if use_final_probs
// Does the computation in two steps
//
// Step 1: Find the value of min_cost, i.e. the minimum cost in the last token
// queue
// (the queue generated by the last frame computed)
// We set both channel_counters->min_int_cost_and_arg_without_final
// and channel_counters->min_int_cost_and_arg_with_final
// One add the final_cost[token.state] before looking for the min
__global__ void get_best_cost_step1_kernel(DeviceParams cst_dev_params,
                                           KernelParams params,
                                           bool use_final_probs) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);
    const int32 main_q_end = channel_counters->prev_main_q_narcs_and_end.y;
    const int32 global_offset = channel_counters->prev_main_q_global_offset;
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(idx, main_q_end) {
      if (idx == 0)
        lane_counters->n_within_lattice_beam =
            0;  // will be used in the next kernel
      const int2 both =
          cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[idx];
      const int token_state = both.x;
      const int token_int_cost = both.y;
      CostType cost = orderedIntToFloat(token_int_cost);
      IntegerCostType int_cost = floatToOrderedInt(cost);
      int32 global_idx = global_offset + idx;
      // We know what is the min cost (without final costs)
      // we just need to have the index of one token with that min cost

      if (use_final_probs) {
        const CostType final_cost =
            cst_dev_params.d_fst_final_costs[token_state];
        IntegerCostType int_cost_with_final =
            floatToOrderedInt(cost + final_cost);
        if (final_cost != cst_dev_params.fst_zero) {
          int2 min_and_arg = {int_cost_with_final,
                              global_idx};  // sort by cost, put it first
          atomicMinI2(&channel_counters->min_int_cost_and_arg_with_final,
                      min_and_arg);
        }
      }
    }
  }
}

// Step2: Now that step1 found the min_cost (with and without final cost)
// If at least one final token (token associated with a final fst state)
// exists in the token queue, AND if use_final_probs is true,
// We can detect all tokens with a cost within [min_cost;min_cost+lattice_beam]
// and list them into d_list_final_tokens_in_main_q
__global__ void get_best_cost_step2_kernel(DeviceParams cst_dev_params,
                                           KernelParams params,
                                           bool use_final_probs) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    const ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);
    const int32 main_q_end = channel_counters->prev_main_q_narcs_and_end.y;
    const int32 global_offset = channel_counters->prev_main_q_global_offset;
    const int2 min_int_cost_and_arg_with_final =
        channel_counters->min_int_cost_and_arg_with_final;
    const int2 min_int_cost_and_arg_without_final =
        channel_counters->min_int_cost_and_arg_without_final;
    bool has_reached_final = (min_int_cost_and_arg_with_final.x != INT_MAX);
    // Use final if we want to use final (use_final_probs is true) and if we
    // found a final state in the token list
    bool compute_final = use_final_probs && has_reached_final;
    IntegerCostType min_cost_to_use =
        compute_final ? min_int_cost_and_arg_with_final.x
                      : min_int_cost_and_arg_without_final.x;

    // if token.cost < lattice_cutoff, that token will belong in the output
    // lattice
    CostType lattice_cutoff =
        orderedIntToFloat(min_cost_to_use) + cst_dev_params.lattice_beam;
    IntegerCostType lattice_int_cutoff = floatToOrderedInt(lattice_cutoff);
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(idx, main_q_end) {
      // First thread of each lane will move the results into lane counters.
      // That's because we never move channel counters back to host,
      // so we move those values to the lane counters, and those lane counters
      // will be moved to host after this kernel
      if (idx == 0) {
        // The lane counters will be copied to host
        lane_counters->min_int_cost_and_arg =
            compute_final ? min_int_cost_and_arg_with_final
                          : min_int_cost_and_arg_without_final;
        lane_counters->has_reached_final = has_reached_final;
      }
      // Looking for a token with its int_cost < lattice_int_cutoff
      const int2 both =
          cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[idx];
      const int32 token_state = both.x;
      int32 token_int_cost = both.y;
      if (compute_final) {
        const CostType final_cost =
            cst_dev_params.d_fst_final_costs[token_state];
        const CostType token_cost = orderedIntToFloat(token_int_cost);
        // final_cost == fst_zero -> this state is not final
        token_int_cost = (final_cost != cst_dev_params.fst_zero)
                             ? floatToOrderedInt(token_cost + final_cost)
                             : INT_MAX;
      }
      if (token_int_cost < lattice_int_cutoff) {
        // That token will be included in the lattice (last frame)
        // save it
        int list_idx = atomicAdd(&lane_counters->n_within_lattice_beam, 1);
        cst_dev_params.h_list_final_tokens_in_main_q.lane(ilane)[list_idx] = {
            global_offset + idx, token_int_cost};
      }
    }
  }
}
__global__ void get_best_cost_step3_kernel(DeviceParams cst_dev_params,
                                           KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *d_lanes_counters =
        cst_dev_params.d_lanes_counters.lane(ilane);
    LaneCounters *h_lanes_counters =
        cst_dev_params.h_lanes_counters.lane(ilane);
    h_lanes_counters->min_int_cost_and_arg =
        d_lanes_counters->min_int_cost_and_arg;
    h_lanes_counters->has_reached_final = d_lanes_counters->has_reached_final;
    h_lanes_counters->n_within_lattice_beam =
        d_lanes_counters->n_within_lattice_beam;
  }
}
// compute_costs_histogram_kernel
// Used in ApplyMaxActiveAndReduceBeam
// Compute the histogram of the token.cost in the main_q
__global__ void compute_costs_histogram_kernel(DeviceParams cst_dev_params,
                                               KernelParams params,
                                               bool use_aux_q) {
  const int nlanes = params.nlanes_used;
  typedef hipcub::BlockHistogram<BinId, KALDI_CUDA_DECODER_1D_BLOCK, 1,
                              KALDI_CUDA_DECODER_HISTO_NBINS + 1>
      BlockHistogram;
  __shared__ typename BlockHistogram::TempStorage temp_storage;
  __shared__ unsigned int smem_histogram[KALDI_CUDA_DECODER_HISTO_NBINS + 1];

  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    const int32 q_end = use_aux_q ? lane_counters->post_expand_aux_q_end
                                  : lane_counters->main_q_narcs_and_end.y;
    bool compute_max_active = lane_counters->compute_max_active;
    if (!compute_max_active) {
      if (q_end <= cst_dev_params.max_active) continue;  // nothing to do
      // Otherwise let's turn max active on for this frame and lane
      lane_counters->compute_max_active = true;
    }

    // Reset local histogram for this lane
    BlockHistogram(temp_storage).InitHistogram(smem_histogram);
    CostType min_histo_cost = lane_counters->min_histo_cost;
    CostType max_histo_cost = lane_counters->max_histo_cost;
    CostType bin_width = lane_counters->histo_bin_width;

    // We have a sync inside the loop, keeping all threads alive
    KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx,
                                                   q_end) {
      const int32 q_idx = block_offset + thread_idx;
      // The last bin is for everything we don't want to count:
      // cost already above the beam, or non-valid tokens
      // It is the default bin
      BinId bin_id[1];
      bin_id[0] = KALDI_CUDA_DECODER_HISTO_NBINS;
      if (q_idx < q_end) {
        IntegerCostType int_cost =
            use_aux_q
                ? cst_dev_params.d_aux_q_state_and_cost.lane(ilane)[q_idx].y
                : cst_dev_params.d_main_q_state_and_cost
                      .channel(ichannel)[q_idx]
                      .y;
        CostType cost = orderedIntToFloat(int_cost);
        CostType extra = cost - min_histo_cost;
	if(extra <= 0.0f) 
		bin_id[0] = 0;
  	else if (extra < max_histo_cost) {
          bin_id[0] = (BinId)__fdiv_rd(extra, bin_width)+1; // +1 because first bin is cost < min_histo_cost
        }
      }
      BlockHistogram(temp_storage).Composite(bin_id, smem_histogram);  // sync
      __syncthreads();  // reusing temp_storage
    }

    // Not using the macros 1D_LOOP because that loop is only within a CTA
    for (int32 bin_id_w = threadIdx.x;
         bin_id_w < KALDI_CUDA_DECODER_HISTO_NBINS;
         bin_id_w += KALDI_CUDA_DECODER_1D_BLOCK) {
      // Writing the local histo to global
      // We don't care about the last bin (cf above)
      int32 s_count = (int32)smem_histogram[bin_id_w];
      atomicAdd(&cst_dev_params.d_histograms.lane(ilane)[bin_id_w], s_count);
    }
    // Making sure we're done reading from smem
    __syncthreads();
  }
}

// update_beam_using_histogram_kernel
// used in ApplyMaxActiveAndReduceBeam
// uses the histogram computed in compute_costs_histogram_kernel
// to find where to cut (where to set the beam)
// to keep only ~max_active_ tokens.
// Important: use only one CTA per lane
__global__ void update_beam_using_histogram_kernel(DeviceParams cst_dev_params,
                                                   KernelParams params,
                                                   bool use_aux_q) {
  typedef hipcub::BlockScan<int, KALDI_CUDA_DECODER_1D_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;

  const int nlanes = params.nlanes_used;
  const int max_active = cst_dev_params.max_active;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    bool compute_max_active = lane_counters->compute_max_active;
    if (!compute_max_active) continue;  // nothing to do
    CostType beam = orderedIntToFloat(lane_counters->int_beam);
    CostType min_histo_cost = lane_counters->min_histo_cost;
    CostType bin_width = lane_counters->histo_bin_width;
    // We now have our histogram of the token costs (computed in the previous
    // kernel)
    // Each thread i is responsible for a bin i, with that bin containing ni
    // tokens.
    // We compute the prefix sum of those ni, ending up for each thread with
    // si=sum[i=1..i](ni)
    // If the thread i detects that si < max_active_ and s[i+1] >= max_active_,
    // then we will cut the beam at
    // the cost of the bin [i+1]
    //
    // Assert : one thread in a CTA is responsible for at most one bin
    // we will not iterate over bins
    assert(KALDI_CUDA_DECODER_HISTO_NBINS < KALDI_CUDA_DECODER_1D_BLOCK);
    int bin_id = threadIdx.x;
    int val = 0;
    if (bin_id < KALDI_CUDA_DECODER_HISTO_NBINS) 
      val = cst_dev_params.d_histograms.lane(ilane)[bin_id];
    
    int prefix_sum;
    BlockScan(temp_storage).ExclusiveSum(val, prefix_sum);

    if (prefix_sum < max_active && (prefix_sum + val) >= max_active) {
      // We found our new beam regarding min_histo_cost
      // Howevever, the current min_cost could be lower than min_histo_cost
      // we need to add that diff to the new beam
      CostType new_beam_for_histo_min_cost = bin_width * bin_id;
      CostType current_min_cost = orderedIntToFloat(lane_counters->min_int_cost);
      CostType new_beam = (min_histo_cost - current_min_cost) + new_beam_for_histo_min_cost;
      IntegerCostType new_int_beam = floatToOrderedInt(new_beam);
      // Saving our new beam for this lane
      lane_counters->int_beam = new_int_beam;
      lane_counters->adaptive_int_beam_with_validity_index.x = new_int_beam;
      lane_counters->int_cutoff = floatToOrderedInt(current_min_cost + new_beam);
    }
  }
}

//
// PostProcessingMainQueue kernels.
// all the following kernels are called when postprocessing a frame
//

// Filling hashmap values with the tokens that we have in the main queue
// We do that because multiple tokens associated with the same FST state
// (but with different arc_idx) can exist in the main_q. We need to detect
// that situation, count them, detect what the min_cost for that FST state is.
// It is done using a hashmap
__global__ void fill_hashmap_with_main_q_kernel(DeviceParams cst_dev_params,
                                                KernelParams params) {
  // Operator for the prefix sum inside the CUDA block
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    ChannelCounters *channel_counters =
        cst_dev_params.d_channels_counters.channel(ichannel);

    const int32 main_q_end = lane_counters->main_q_narcs_and_end.y;
    int32 min_int_cost = lane_counters->min_int_cost;
    CostType min_cost = orderedIntToFloat(min_int_cost);
    const int32 global_offset = lane_counters->main_q_global_offset;
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(main_q_idx, main_q_end) {
      // Position of considered token in the main_q
      if (main_q_idx < main_q_end) {
        int2 both = cst_dev_params.d_main_q_state_and_cost.channel(
            ichannel)[main_q_idx];
        StateId token_state = both.x;
        IntegerCostType token_int_cost = both.y;
        if (min_int_cost == token_int_cost) {
          // remove offset = min_cost, set it to 0 explicitely
          token_int_cost = floatToOrderedInt(0.0f);
          channel_counters->min_int_cost_and_arg_without_final = {
              token_int_cost, global_offset + main_q_idx};
          lane_counters->prev_arg_min_int_cost = main_q_idx;
        } else {
          // remove offset = min_cost
          CostType token_cost = orderedIntToFloat(token_int_cost) - min_cost;
          token_int_cost = floatToOrderedInt(token_cost);
        }
        int local_idx, hash_idx;
        hashmap_insert_or_aggregate(cst_dev_params.d_hashmap_values.lane(ilane),
                                    token_state, token_int_cost, main_q_idx,
                                    cst_dev_params.hashmap_capacity, &local_idx,
                                    &hash_idx);
        cst_dev_params.d_main_q_n_extra_prev_tokens_local_idx.lane(
            ilane)[main_q_idx] = local_idx;
        cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[main_q_idx].y =
            token_int_cost;
        // If we have the min, saving its index for get best cost and the min
        // cost estimate of the next frame

        // Saving where that token.state ended up in the hashmap
        // false = this token is not the representative of this state
        // We will update representing_state once we know more (in the next
        // kernel)
        // We first need to add all tokens to the hashmap. Which will be the
        // case when
        // this kernel returns.
        SetFSTStateHashIndex(
            hash_idx, false,
            &cst_dev_params.d_main_q_state_hash_idx.lane(ilane)[main_q_idx]);
      }

      if (main_q_idx == 0) {
        lane_counters->int_cutoff = floatToOrderedInt(
            orderedIntToFloat(lane_counters->int_cutoff) - min_cost);
      }
    }
  }
}

// preprocess_and_list_extra_prev_tokens_kernel_step[i] kernels
// Called in PostProcessingMainQueue
// They do two things:
// - do the "emitting preprocessing". I.e. doing the preprocessing necessary for
// the future ExpandArcsEmitting that may be done next (if the current frame is
// not the last one)
// It consists of filling the d_main_q_degrees_prefix_sum of the emitting arc
// degrees of the tokens + setting d_main_q_arc_offsets
// - when we have multiple tokens associated with the same FST state S, we will
// list them in d_main_q_extra_prev_tokens. We need to know where to put them in
// that array,
// so we'll compute a prefix_sum also to compute those indexes. We'll then save
// the location of each extra tokens list (its offset and size in
// d_main_q_extra_prev_tokens),
// and save it into d_main_q_info for later lattice processing
//
// First step : Reading the hashmap, detecting which token is representative for
// each FST state, which is decided by fill_hashmap_with_main_q_kernel()
// (we pick one of the best ones, with the best ones being the ones with the
// lowest cost)
// this representative will be responsible for K tokens, with K being the number
// of tokens associated with that FST state. We only considers the cases where K
// > 1,
// because if K == 1, then we will not store that token in the special list
// d_main_q_extra_prev_tokens
// Each representative is also the only token that will propagate emitting arcs
// for that FST state. Because a representative has the min_cost for that FST
// state, it is enough to only propagate
// that one
// Each representative counts the number of emitting arcs it is responsible for,
// and we will compute the prefix sum of the arc degrees
__global__ void emitting_preprocess_and_list_extra_prev_tokens_step1_kernel(
    DeviceParams cst_dev_params, KernelParams params) {
  // Operator for the prefix sum inside the CUDA block
  typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_1D_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage sh_temp_storage;
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters =
        cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 main_q_end = lane_counters->main_q_narcs_and_end.y;
    // Final cutoff from last ExpandArc execution
    // The cutoff can have decreased since moving tokens to the main_q
    // min_cost cannot be lower than before (we only did non-emitting phases
    // since then)
    // but the adaptive beam may have lowered the beam
    const IntegerCostType int_cutoff = lane_counters->int_cutoff;
    // Keeping all threads in CTA alive
    // We'll __syncthreads()
    KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx,
                                                   main_q_end) {
      // We'll take care of the token at index main_q_idx
      const int32 main_q_idx = block_offset + thread_idx;
      const int32 ichannel = lane_counters->channel_to_compute;
      // If that token is the representative of its FST state (token.next_state)
      // The representative of a FST state is the token with the lowest
      // token.cost for that FST state
      // If multiple tokens have token1.cost == token2.cost ==
      // min_cost_for_that_state, then one is picked (first come first serve,
      // was done in fill_hashmap_with_main_q_kernel)
      bool representing_state = false;
      // Number of emitting arcs for that token
      // Only the token representative of that FST state can have degree > 0
      int32 degree = 0;
      // If that token is representative of a FST state S,
      // and if multiple tokens are associated with that state S,
      // then n_extra_prev_token will contain their count
      int32 n_extra_prev_token = 0;
      if (main_q_idx < main_q_end) {
        int2 both = cst_dev_params.d_main_q_state_and_cost.channel(
            ichannel)[main_q_idx];
        StateId token_state = both.x;
        IntegerCostType token_int_cost = both.y;
        // Loading info about token.next_state. Is there multiple tokens for
        // that state ?
        // How many ? What's the min token.cost for that state ?
        int32 hash_idx;    // we saved the hash_idx after inserting
        bool bool_buffer;  // will always be false. We just need it to call the
                           // function
        GetFSTStateHashIndex(
            cst_dev_params.d_main_q_state_hash_idx.lane(ilane)[main_q_idx],
            &hash_idx, &bool_buffer);
        HashmapValueT h_val =
            cst_dev_params.d_hashmap_values.lane(ilane)[hash_idx];
        // Token index of one of the token which the lowest token.cost for that
        // state
        uint32_t state_best_int_cost_argmin;
	GetArgFromPackedArgminUInt64(h_val.min_and_argmin_int_cost_u64, &state_best_int_cost_argmin);

        // Checking if we're the representative of that state
        representing_state = (main_q_idx == state_best_int_cost_argmin);
        // Saving the hash_idx of that fst state + if we're responsible for that
        // state
        SetFSTStateHashIndex(
            hash_idx, representing_state,
            &cst_dev_params.d_main_q_state_hash_idx.lane(ilane)[main_q_idx]);

        // One of the best token for that state will represent that state in the
        // next frame
        if (representing_state) {
          if (token_int_cost < int_cutoff) {
            // Next step is emitting (next frame), using emitting offsets
            const int32 start = cst_dev_params.d_arc_e_offsets[token_state];
            const int32 end = cst_dev_params.d_arc_e_offsets[token_state + 1];
            degree = end - start;
            // Saving the start offset for the expand kernel
            // avoid a new random memory access
            cst_dev_params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx] =
                start;

	    // Saving best cost with final cost, to compute the final_extra_cost
	    // It seems like ~5% of all states are final, so the following atomic may be fine
	    // if necessary, we could first reduce locally at the CTA level
	    const CostType final_cost =
		    cst_dev_params.d_fst_final_costs[token_state];
	    if(final_cost != cst_dev_params.fst_zero) {
		    IntegerCostType token_int_cost_with_final = floatToOrderedInt(orderedIntToFloat(token_int_cost) + final_cost);
		    IntegerCostType int_relative_cost = token_int_cost_with_final; // - 0.0f, the min_cost was reset to 0.0f
		    atomicMin(&lane_counters->int_relative_cost, int_relative_cost);
	    }
          }
          // If that FST state has only one token associated to it, we store
          // that token directly in
          // d_main_q_info (its original place)
          // We only move it into the d_main_q_extra_prev_tokens list if
          // multiple tokens are associated to that state
          n_extra_prev_token = (h_val.count > 1) ? (h_val.count) : 0;
        }
      }

      // Computing a local prefix sum inside that CUDA block
      // Others kernels will take care of adding the necessary offset to those
      // local prefix sums
      int2 zeroi2 = {0, 0};
      int2 vali2 = {degree, n_extra_prev_token};
      int2 aggi2;
      BlockScan(sh_temp_storage)
          .ExclusiveScan(vali2, aggi2, zeroi2, PlusPlus());
      int32 degree_local_prefix_sum = aggi2.x;
      int32 n_extra_prev_token_prefix_sum = aggi2.y;

      if (main_q_idx < main_q_end) {
        // This is not the final global prefix sum
        // Other kernels will add the necessary offset
        cst_dev_params.d_main_q_degrees_prefix_sum.channel(
            ichannel)[main_q_idx] = degree_local_prefix_sum;
        cst_dev_params.d_main_q_extra_prev_tokens_prefix_sum.lane(
            ilane)[main_q_idx] = n_extra_prev_token_prefix_sum;
      }

      if (KALDI_CUDA_DECODER_IS_LAST_1D_THREAD()) {
        // Saving the local sum of degrees of that CUDA block
        // That's necessary to compute the global offset of that CUDA block,
        // and that offset is what we need to transform the local prefix sum
        // into a global prefix sum
        const int local_sum_index = block_offset / KALDI_CUDA_DECODER_1D_BLOCK;
        // the prefix sum was exclusive, adding missing value
        const int degree_inclusive_sum = degree_local_prefix_sum + degree;
        const int n_extra_prev_tokens_inclusive_sum =
            n_extra_prev_token_prefix_sum + n_extra_prev_token;
        cst_dev_params.d_main_q_block_sums_prefix_sum.lane(
            ilane)[local_sum_index] = {degree_inclusive_sum,
                                       n_extra_prev_tokens_inclusive_sum};
      }

      // Synchronization because:
      // - we may need to reuse sh_temp_storage if the for loop iterates (cf
      // CUB's doc)
      __syncthreads();
    }
  }
}

// In step1, we've computed the local (CTA-wide) prefix sums. We also have the
// local sums of each individual CTAs
// In this kernel, we will compute the offset of each CTA in the global prefix
// sum. We will then add those offsets in step3
// Only one CTA / lane
__global__ void emitting_preprocess_and_list_extra_prev_tokens_step2_kernel(
    DeviceParams cst_dev_params, KernelParams params) {
  typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_1D_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage sh_temp_storage;
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int main_q_end = lane_counters->main_q_narcs_and_end.y;
    const int ntiles = KALDI_CUDA_DECODER_DIV_ROUND_UP(
        main_q_end, KALDI_CUDA_DECODER_1D_BLOCK);
    // Using block_offset loop to keep entire CTA alive (we're going to use
    // __syncthreads in CUB)
    int2 sum_so_far = {0, 0};
    KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(offset, thread_idx, ntiles) {
      const int32 itile = offset + thread_idx;
      const int2 zeroi2 = {0, 0};
      const int2 val =
          (itile < ntiles)
              ? cst_dev_params.d_main_q_block_sums_prefix_sum.lane(ilane)[itile]
              : zeroi2;

      int2 prefix_sum, sum;
      BlockScan(sh_temp_storage)
          .ExclusiveScan(val, prefix_sum, zeroi2, PlusPlus(), sum);
      PlusPlus pp;
      prefix_sum = pp(prefix_sum, sum_so_far);
      sum_so_far = pp(sum_so_far, sum);
      if (itile < ntiles) {
        cst_dev_params.d_main_q_block_sums_prefix_sum.lane(ilane)[itile] =
            prefix_sum;
      }
      if (itile == (ntiles - 1)) {
        const int32 total_narcs = prefix_sum.x + val.x;
        const int32 total_n_extra_prev_tokens = prefix_sum.y + val.y;
        lane_counters->main_q_narcs_and_end.x = total_narcs;
        lane_counters->main_q_n_extra_prev_tokens = total_n_extra_prev_tokens;
        assert(total_n_extra_prev_tokens >= 0 &&
               total_n_extra_prev_tokens <= main_q_end);
      }
    }
  }
}

// Step3: Uses the CTA offsets computed in step2 to transform the CTA-wide
// prefix sums to global prefix sums
// The representative of each FST states saves into the hashmap the location of
// the extra_prev_tokens of that state
// in d_main_q_extra_prev_tokens. That way each extra tokens will know where to
// write itself in the next kernel.
__global__ void emitting_preprocess_and_list_extra_prev_tokens_step3_kernel(
		DeviceParams cst_dev_params, KernelParams params) {
	const int nlanes = params.nlanes_used;
	KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
		const LaneCounters *lane_counters =
			cst_dev_params.d_lanes_counters.lane(ilane);
		const int32 ichannel = lane_counters->channel_to_compute;
		const int main_q_end = lane_counters->main_q_narcs_and_end.y;
		KALDI_CUDA_DECODER_1D_KERNEL_LOOP(main_q_idx, main_q_end) {
			const int32 local_sum_idx = main_q_idx / KALDI_CUDA_DECODER_1D_BLOCK;
			const int2 local_sum_offset =
				cst_dev_params.d_main_q_block_sums_prefix_sum.lane(
						ilane)[local_sum_idx];
			cst_dev_params.d_main_q_degrees_prefix_sum.channel(
					ichannel)[main_q_idx] += local_sum_offset.x;
			int extra_prev_tokens_offset =
				cst_dev_params.d_main_q_extra_prev_tokens_prefix_sum.lane(
						ilane)[main_q_idx] +
				local_sum_offset.y;
			// Loading the hash index associate with token.state
			// If representative, store the location of the extra prev tokens list for
			// that state in the hashmap
			bool is_representative;
			int32 hash_idx;
			GetFSTStateHashIndex(
					cst_dev_params.d_main_q_state_hash_idx.lane(ilane)[main_q_idx],
					&hash_idx, &is_representative);
                        if (is_representative) {
                          HashmapValueT &val =
                              cst_dev_params.d_hashmap_values.lane(
                                  ilane)[hash_idx];
                          uint32_t min;
                          GetMinFromPackedArgminUInt64(
                              val.min_and_argmin_int_cost_u64, &min);
                          unsigned long long new_pack;
                          PackArgminInUInt64(min, extra_prev_tokens_offset,
                                             &new_pack);
                          val.min_and_argmin_int_cost_u64 = new_pack;
                        }
		}
	}
}

// Step4: We now know where to store our extra prev tokens in
// d_main_q_extra_prev_tokens.
// We will now move the tokens that need to be moved (when multiple tokens are
// associated to the same FST state)
// into d_main_q_extra_prev_tokens. In d_main_q_info, we will store the location
// of that list [offset,size]
// so that when backtracking, when we read d_main_q_info[token_idx], we know
// where to look to have the list
// of the same-state tokens
// It is the last step of the
// emitting_preprocess_and_list_extra_prev_tokens_step[i]_kernel pipeline
__global__ void emitting_preprocess_and_list_extra_prev_tokens_step4_kernel(
    DeviceParams cst_dev_params, KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    const LaneCounters *lane_counters =
        cst_dev_params.d_lanes_counters.lane(ilane);
    const int32 ichannel = lane_counters->channel_to_compute;
    const int main_q_end = lane_counters->main_q_narcs_and_end.y;
    // Previous frames have filled d_main_q_extra_prev_tokens.
    // d_main_q_extra_prev_tokens was then flushed to host. We want to set the
    // global
    // (global in the sense "for all frames") offset on where to read it the
    // h_all_tokens_extra_prev_tokens_ on host.
    // adding the main_q_extra_prev_tokens_global_offset for that
    const int prev_global_idx =
        lane_counters->main_q_extra_prev_tokens_global_offset;
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(main_q_idx, main_q_end) {
      // We'll take care of token at main_q_idx
      // Loading hashmap information about token.state
      bool is_representative;
      int32 hash_idx;
      GetFSTStateHashIndex(
          cst_dev_params.d_main_q_state_hash_idx.lane(ilane)[main_q_idx],
          &hash_idx, &is_representative);

      HashmapValueT val = cst_dev_params.d_hashmap_values.lane(ilane)[hash_idx];
      // How many tokens are associated with that fst state token.state
      int same_count = val.count;
      bool must_move_to_extra_prev_tokens = (same_count > 1);
      if (must_move_to_extra_prev_tokens) {
        // Moving to the extra_prev_tokens list.
        // Some of those tokens have an extra cost (compared to the best cost
        // for that FST state)
        // Generating and saving that extra cost. We will use it when generating
        // the lattice.
        CostType token_cost = orderedIntToFloat(
            cst_dev_params.d_main_q_state_and_cost.channel(ichannel)[main_q_idx]
                .y);
	uint32_t best_int_cost;
        // Where to write this state list in d_main_q_extra_prev_tokens
	uint32_t extra_prev_tokens_offset;
	unsigned long long pack = val.min_and_argmin_int_cost_u64;
	GetMinFromPackedArgminUInt64(pack, &best_int_cost);
	GetArgFromPackedArgminUInt64(pack, &extra_prev_tokens_offset);
        CostType best_cost = orderedIntToFloat((int)best_int_cost);
        CostType extra_cost = token_cost - best_cost;
	assert(!is_representative || extra_cost == 0.0f);
        // Loading the token to be moved
        InfoToken inf_tok =
            cst_dev_params.d_main_q_info.lane(ilane)[main_q_idx];
        CostType acoustic_cost =
            cst_dev_params.d_main_q_acoustic_cost.lane(ilane)[main_q_idx];
        // Place of that specific token in the extra_prev_tokens sublist of that
        // specific FST state
        int32 local_idx =
            cst_dev_params.d_main_q_n_extra_prev_tokens_local_idx.lane(
                ilane)[main_q_idx];
        // Saving the location of the extra prev tokens for that state into that
        // InfoToken
        SetSameFSTStateTokensList(
            prev_global_idx + extra_prev_tokens_offset, same_count,
            &cst_dev_params.d_main_q_info.lane(ilane)[main_q_idx]);
        // Where to write this token in d_main_q_extra_prev_tokens
        int32 list_idx = extra_prev_tokens_offset + local_idx;
        // Moving token. Also saving extra_cost
        cst_dev_params.d_main_q_extra_prev_tokens.lane(ilane)[list_idx] =
            inf_tok;
        cst_dev_params.d_main_q_extra_and_acoustic_cost.lane(
            ilane)[list_idx] = {extra_cost, acoustic_cost};
        assert(inf_tok.prev_token >= (lane_counters->main_q_global_offset -
                                      cst_dev_params.main_q_capacity) &&
               inf_tok.prev_token <=
                   (lane_counters->main_q_global_offset + main_q_end));
      }
    }
  }
}

// Clear the hashmaps after use
// Each element in the map has a representative in the main_q
// Everyone of those representatives has the responsability to reset their
// corresponding value in the hashmap
// Once this kernel returns, the hashmaps are cleared
__global__ void clear_hashmap_kernel(DeviceParams cst_dev_params,
                                     KernelParams params) {
  const int nlanes = params.nlanes_used;
  KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
    LaneCounters *lane_counters = cst_dev_params.d_lanes_counters.lane(ilane);
    const int main_q_end = lane_counters->main_q_narcs_and_end.y;
    KALDI_CUDA_DECODER_1D_KERNEL_LOOP(main_q_idx, main_q_end) {
      bool is_representative;
      int32 hash_idx;
      GetFSTStateHashIndex(
          cst_dev_params.d_main_q_state_hash_idx.lane(ilane)[main_q_idx],
          &hash_idx, &is_representative);
      // Representative owns a state. Each representative resets its associated
      // token.state
      // in the hashmap
      if (is_representative) {
        cst_dev_params.d_hashmap_values.lane(ilane)[hash_idx] =
            KALDI_CUDA_DECODER_HASHMAP_NO_VAL;  // clear
      }
    }
  }
}

// Kernels wrappers

void SaveChannelsStateFromLanesKernel(const dim3 &grid, const dim3 &block,
                                      const hipStream_t &st,
                                      const DeviceParams &cst_dev_params,
                                      const KernelParams &kernel_params) {
  save_channels_state_from_lanes_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                                kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void LoadChannelsStateInLanesKernel(const dim3 &grid, const dim3 &block,
                                    const hipStream_t &st,
                                    const DeviceParams &cst_dev_params,
                                    const KernelParams &kernel_params) {
  load_channels_state_in_lanes_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                              kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void InitDecodingOnDeviceKernel(const dim3 &grid, const dim3 &block,
                                const hipStream_t &st,
                                const DeviceParams &cst_dev_params,
                                const KernelParams &kernel_params) {
  init_decoding_on_device_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                         kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void InitializeInitialLaneKernel(const dim3 &grid, const dim3 &block,
                                 const hipStream_t &st,
                                 const DeviceParams &cst_dev_params) {
  initialize_initial_lane_kernel<<<grid, block, 0, st>>>(cst_dev_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void ResetForFrameAndEstimateCutoffKernel(const dim3 &grid, const dim3 &block,
                                          const hipStream_t &st,
                                          const DeviceParams &cst_dev_params,
                                          const KernelParams &kernel_params) {
  reset_for_frame_and_estimate_cutoff_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params);
}

template <bool IS_EMITTING>
void ExpandArcsKernel(const dim3 &grid, const dim3 &block,
                      const hipStream_t &st,
                      const DeviceParams &cst_dev_params,
                      const KernelParams &kernel_params) {
  expand_arcs_kernel<IS_EMITTING><<<grid, block, 0, st>>>(cst_dev_params,
                                                          kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

template <bool IS_EMITTING>
void PostExpandKernel(const dim3 &grid, const dim3 &block,
                      const hipStream_t &st,
                      const DeviceParams &cst_dev_params,
                      const KernelParams &kernel_params) {
  post_expand_kernel<IS_EMITTING><<<grid, block, 0, st>>>(cst_dev_params,
                                                          kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void PostContractAndPreprocessKernel(const dim3 &grid, const dim3 &block,
                                     const hipStream_t &st,
                                     const DeviceParams &cst_dev_params,
                                     const KernelParams &kernel_params) {
  post_contract_and_preprocess_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                              kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void NonEmittingPreprocessAndContractKernel(const dim3 &grid, const dim3 &block,
                                            const hipStream_t &st,
                                            const DeviceParams &cst_dev_params,
                                            const KernelParams &kernel_params) {
  nonemitting_preprocess_and_contract_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void FillHashmapWithMainQKernel(const dim3 &grid, const dim3 &block,
                                const hipStream_t &st,
                                const DeviceParams &cst_dev_params,
                                const KernelParams &kernel_params) {
  fill_hashmap_with_main_q_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                          kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void EmittingPreprocessAndListExtraPrevTokensStep1Kernel(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &kernel_params) {
  emitting_preprocess_and_list_extra_prev_tokens_step1_kernel<<<grid, block, 0,
                                                                st>>>(
      cst_dev_params, kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void EmittingPreprocessAndListExtraPrevTokensStep2Kernel(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &kernel_params) {
  emitting_preprocess_and_list_extra_prev_tokens_step2_kernel<<<grid, block, 0,
                                                                st>>>(
      cst_dev_params, kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void EmittingPreprocessAndListExtraPrevTokensStep3Kernel(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &kernel_params) {
  emitting_preprocess_and_list_extra_prev_tokens_step3_kernel<<<grid, block, 0,
                                                                st>>>(
      cst_dev_params, kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void EmittingPreprocessAndListExtraPrevTokensStep4Kernel(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &kernel_params) {
  emitting_preprocess_and_list_extra_prev_tokens_step4_kernel<<<grid, block, 0,
                                                                st>>>(
      cst_dev_params, kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void ComputeLaneOffsetsKernel(const dim3 &grid, const dim3 &block,
                              const hipStream_t &st,
                              const DeviceParams &cst_dev_params,
                              const KernelParams &kernel_params) {
  compute_lane_offsets_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                      kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

template <typename T>
void ConcatenateLanesDataKernel(const dim3 &grid, const dim3 &block,
                                const hipStream_t &st,
                                const DeviceParams &cst_dev_params,
                                const KernelParams &kernel_params,
                                const LaneMatrixView<T> &src, T *concat,
                                int32 *lane_offsets) {
  concatenate_lanes_data_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params, src, concat, lane_offsets);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void InitHashmapKernel(const dim3 &grid, const dim3 &block,
                       const hipStream_t &st,
                       const DeviceParams &cst_dev_params) {
  init_hashmap_kernel<<<grid, block, 0, st>>>(cst_dev_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void ClearHashmapKernel(const dim3 &grid, const dim3 &block,
                        const hipStream_t &st,
                        const DeviceParams &cst_dev_params,
                        const KernelParams &kernel_params) {
  clear_hashmap_kernel<<<grid, block, 0, st>>>(cst_dev_params, kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void ComputeCostsHistogramKernel(const dim3 &grid, const dim3 &block,
                                 const hipStream_t &st,
                                 const DeviceParams &cst_dev_params,
                                 const KernelParams &kernel_params,
                                 bool use_aux_q) {
  compute_costs_histogram_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params, use_aux_q);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void UpdateBeamUsingHistogramKernel(const dim3 &grid, const dim3 &block,
                                    const hipStream_t &st,
                                    const DeviceParams &cst_dev_params,
                                    const KernelParams &kernel_params,
                                    bool use_aux_q) {
  update_beam_using_histogram_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params, use_aux_q);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void FinalizeProcessNonEmittingKernel(const dim3 &grid, const dim3 &block,
                                      const hipStream_t &st,
                                      const DeviceParams &cst_dev_params,
                                      const KernelParams &kernel_params) {
  finalize_process_non_emitting_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                               kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void GetBestCostStep1Kernel(const dim3 &grid, const dim3 &block,
                            const hipStream_t &st,
                            const DeviceParams &cst_dev_params,
                            const KernelParams &kernel_params, bool isfinal) {
  get_best_cost_step1_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params, isfinal);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void GetBestCostStep2Kernel(const dim3 &grid, const dim3 &block,
                            const hipStream_t &st,
                            const DeviceParams &cst_dev_params,
                            const KernelParams &kernel_params, bool isfinal) {
  get_best_cost_step2_kernel<<<grid, block, 0, st>>>(
      cst_dev_params, kernel_params, isfinal);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

void GetBestCostStep3Kernel(const dim3 &grid, const dim3 &block,
                            const hipStream_t &st,
                            const DeviceParams &cst_dev_params,
                            const KernelParams &kernel_params) {
  get_best_cost_step3_kernel<<<grid, block, 0, st>>>(cst_dev_params,
                                                     kernel_params);
  KALDI_DECODER_CUDA_CHECK_ERROR();
}

template void ExpandArcsKernel<true>(const dim3 &grid, const dim3 &block,
                                     const hipStream_t &st,
                                     const DeviceParams &cst_dev_params,
                                     const KernelParams &params);
template void ExpandArcsKernel<false>(const dim3 &grid, const dim3 &block,
                                      const hipStream_t &st,
                                      const DeviceParams &cst_dev_params,
                                      const KernelParams &params);
template void PostExpandKernel<true>(const dim3 &grid, const dim3 &block,
                                     const hipStream_t &st,
                                     const DeviceParams &cst_dev_params,
                                     const KernelParams &params);
template void PostExpandKernel<false>(const dim3 &grid, const dim3 &block,
                                      const hipStream_t &st,
                                      const DeviceParams &cst_dev_params,
                                      const KernelParams &params);

template void ConcatenateLanesDataKernel<InfoToken>(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &params,
    const LaneMatrixView<InfoToken> &src, InfoToken *concat,
    int32 *lane_offsets);

template void ConcatenateLanesDataKernel<CostType>(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &params,
    const LaneMatrixView<CostType> &src, CostType *concat, int32 *lane_offsets);

template void ConcatenateLanesDataKernel<float2>(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &params,
    const LaneMatrixView<float2> &src, float2 *concat, int32 *lane_offsets);

template void ConcatenateLanesDataKernel<int32>(
    const dim3 &grid, const dim3 &block, const hipStream_t &st,
    const DeviceParams &cst_dev_params, const KernelParams &params,
    const LaneMatrixView<int32> &src, int32 *concat, int32 *lane_offsets);

}  // end namespace cuda_decoder
}  // end namespace kaldi
