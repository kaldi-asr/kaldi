#include "hip/hip_runtime.h"
// cudafeat/feature-online-cmvn-cuda.cu
//
// Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
// Justin Luitjens
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hipcub/hipcub.hpp>
#include "cudafeat/feature-online-cmvn-cuda.h"
#include "cudamatrix/cu-matrix.h"
#include "cudamatrix/cu-vector.h"

__host__ __device__ inline float2 operator-(const float2 &a, const float2 &b) {
  float2 retval;
  retval.x = a.x - b.x;
  retval.y = a.y - b.y;
  return retval;
}
__host__ __device__ inline float2 operator+(const float2 &a, const float2 &b) {
  float2 retval;
  retval.x = a.x + b.x;
  retval.y = a.y + b.y;
  return retval;
}

#if __CUDA_ARCH__ == 750
__launch_bounds__ (1024, 1)
#else
__launch_bounds__ (1024, 2)
#endif
__global__ void compute_cmvn_stats_kernel(const float *data, int32_t ldd,
                                          int32_t num_frames, int32_t feat_dim,
                                          float *stats, int32_t lds) {
  typedef hipcub::BlockScan<float2, 1024> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;

  int32_t feat = blockIdx.x;

  float2 running_sum = {0.0f, 0.0f};
  // for each frame, keep threads alive for cub
  for (int32_t r = 0; r < num_frames; r += blockDim.x) {
    int32_t rid = r + threadIdx.x;

    float val = 0.0f;

    if (rid < num_frames) {
      // uncoalesced, could transpose data or do some shared memory swizzling...
      val = data[rid * ldd + feat];
    }

    float2 sum = {val, val * val};  // this elements value and value squared

    float2 psum;   // row prefix sum
    float2 total;  // total count
    BlockScan(temp_storage).InclusiveSum(sum, psum, total);

    // offset by running sum
    psum = psum + running_sum;
    // increase running sum by new total
    running_sum = running_sum + total;

    // un-coalesced
    if (rid < num_frames) {
      reinterpret_cast<float2 *>(&stats[rid * lds])[feat] = psum;
    }
  }
}

__global__ void apply_cmvn_kernel(
    int32_t cmvn_window, bool var_norm, bool mean_norm, const float *feat_in,
    int32_t ldi, int32_t num_rows, int32_t num_cols,
    const float *__restrict__ stats, int32_t lds,
    const float *__restrict__ global_stats, int32_t ldg, int32_t global_frames,
    const float *__restrict__ speaker_stats, int32_t ldss,
    int32_t speaker_frames, float *feat_out, int32_t ldo) {
  int32_t r = blockIdx.x;

  for (int c = threadIdx.x; c < num_cols; c += blockDim.x) {
    float2 frame_stats =
        reinterpret_cast<const float2 *__restrict__>(&stats[r * lds])[c];

    float val = feat_in[r * ldi + c];

    float window_length = min(r + 1, cmvn_window);

    // we have to subtract row r-cmvn_window stats
    if (r >= cmvn_window) {
      // window starting row
      int32_t o = r - cmvn_window;

      // stats at the start row of the window that must be removed
      float2 ostats =
          reinterpret_cast<const float2 *__restrict__>(&stats[o * lds])[c];

      // remove start of the window stats
      frame_stats = frame_stats - ostats;
    }

    // Smooth stats by speaker frames if necessary
    float smooth_frames = cmvn_window - window_length;
    if (smooth_frames > 0 && speaker_frames > 0) {
      float count_from_speaker = min(smooth_frames, (float)speaker_frames);
      float speaker_count = speaker_stats[num_cols];

      if (count_from_speaker > 0.0) {
        float alpha = count_from_speaker / speaker_count;

        frame_stats.x += alpha * speaker_stats[c];         // update mean
        frame_stats.y += alpha * speaker_stats[ldss + c];  // update variance
        window_length += alpha * speaker_count;  // update window length

        // recompute smooth frames now that we have speaker stats
        smooth_frames = cmvn_window - window_length;
      }
    }

    // Smooth stats by global frames if necessary
    if (smooth_frames > 0 && global_frames > 0) {
      float count_from_global = min(smooth_frames, (float)global_frames);
      float global_count = global_stats[num_cols];

      if (count_from_global > 0.0) {
        float alpha = count_from_global / global_count;

        frame_stats.x += alpha * global_stats[c];        // update mean
        frame_stats.y += alpha * global_stats[ldg + c];  // update variance
        window_length += alpha * global_count;           // update window length
      }
    }

    float mean = frame_stats.x / window_length;
    float var = frame_stats.y / window_length - mean * mean;

    float floor = 1e-20;
    if (var < floor)  // avoid dividing by zero
      var = floor;

    if (!var_norm) {
      // skip variance normalization
      var = 1.0f;
    }
    if (!mean_norm) {
      assert(false);
      // skip mean normalization
      mean = 0.0f;
    }

    // shift by mean and scale by variance
    feat_out[r * ldo + c] = (val - mean) / sqrtf(var);
  }
}

namespace kaldi {

void CudaOnlineCmvn::ComputeFeatures(const CuMatrixBase<BaseFloat> &feats_in,
                               CuMatrix<BaseFloat> *feats_out) {
  int32_t num_frames = feats_in.NumRows();
  int32_t feat_dim = feats_in.NumCols();
  feats_out->Resize(num_frames, feat_dim, kUndefined);

  CuMatrix<float> stats(num_frames, feat_dim * 2, kUndefined);

  int threads = 1024;
  int blocks = feat_dim;

  // compute windowed sum/sum2 prefix sum along column of feats
  compute_cmvn_stats_kernel<<<blocks, threads>>>(
      feats_in.Data(), feats_in.Stride(), num_frames, feat_dim, stats.Data(),
      stats.Stride());
  CU_SAFE_CALL(hipGetLastError());

  threads = (feat_dim + 31) / 32 * 32;  // round up to 32 threads
  if (threads > 1024) threads = 1024;

  const CuMatrix<float> &gstats = cmvn_state_.global_cmvn_stats;
  const CuMatrix<float> &sstats = cmvn_state_.speaker_cmvn_stats;

  int global_frames = opts_.global_frames;
  int speaker_frames = opts_.speaker_frames;

  if (gstats.NumRows() == 0) global_frames = 0;
  if (sstats.NumRows() == 0) speaker_frames = 0;

  // apply cmvn
  apply_cmvn_kernel<<<num_frames, threads>>>(
      opts_.cmn_window, opts_.normalize_variance, opts_.normalize_mean,
      feats_in.Data(), feats_in.Stride(), num_frames, feat_dim, stats.Data(),
      stats.Stride(), gstats.Data(), gstats.Stride(), global_frames,
      sstats.Data(), sstats.Stride(), speaker_frames, feats_out->Data(),
      feats_out->Stride());
  CU_SAFE_CALL(hipGetLastError());
}
}
