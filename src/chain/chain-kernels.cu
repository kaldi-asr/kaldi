#include "hip/hip_runtime.h"
// chain/chain-kernels.cu

// Copyright  2015-2019  Johns Hopkins University (author: Daniel Povey)


// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.


#include <cfloat>
#include "chain/chain-kernels-ansi.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 200
#error - Kaldi no longer supports CC1.x devices. Please use a newer GPU or \
         configure with --use-cuda=no (this will disable the use of GPU).
#endif


#ifdef __HIPCC__
#if ( __CUDACC_VER_MAJOR__ >= 8 ) && ( !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600 )
// native implementation available
#else
#if __CUDA_ARCH__ >= 600
#error using CAS implementation of double atomicAdd
#endif
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*) address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif
#endif


template <typename Real>
__device__ inline void atomic_add(Real* address, Real value) {
  atomicAdd(address, value);
}

template <typename Real>
__device__ inline void atomic_add_thresholded(Real* address, Real value) {
  // This function uses a randomized algorithm to only do atomic adds for values
  // >=n a threshold, and if it's below the threshold, randomly add the
  // threshold itself with probability (value / threshold).  This preserves
  // expectations.  Note: we assume that value >= 0.

  // kThresholdingPowerOfTwo is defined in chain-datastruct.h; it defines
  // the threshold for randomized posterior pruning.
  const Real threshold = 1.0 / (1 << kThresholdingPowerOfTwo);
  if (value >= threshold) {
    atomic_add(address, value);
  } else {
    // The intention here is to do:
    // with probability(value / threshold), do:
    //   atomic_add(address, threshold);
    // We use the least significant bits of the value as a source of
    // randomness.  It would probably be more efficient to extract these
    // random bits directly from the float, but I don't want to have to
    // deal with endian-ness issues.
    //
    // below, x is a fixed-point representation of (value / threshold); it would
    // be 16777216 == 2^24 if value == threshold and 0 if value == 0.  We choose
    // the power 24 because that's the number of binary digits in the mantissa
    // in IEEE single precision floating point.
    // Note: we parenthesize the expression like this so that the
    // denominator can be precomputed as a constant expression.
    int32_cuda x = value / (threshold / (1 << 24));
    // in the line below, the expression (x >> 12) is a representation of (value /
    // threshold) between 0 and 4096, with 4096 representing (value / threshold ==
    // 1), while (x & 4095) is treated as a pseudorandom number between 0 and 4095.
    if ((x >> 12) > (x & 4095))
      atomic_add(address, threshold);
  }
}

// one iteration of the forward computation in the 'tombstone' CTC HMM computation.
// The grid y determines which HMM-state we handle.  [put this in the grid because
// HMM-states don't all take the same amount of time in the backwards direction, and it's
// better for scheduling to have them at the outer level.]
// The block x and grid x determine which sequence (0 ... num_sequences - 1) we handle;
// note that num_sequences == the number of elements in the minibatch, and we
// insist they all have the same number of time steps.
// note: 'probs' is indexed by sequence-index + (pdf-index * prob_stride).
__global__
static void _cuda_chain_hmm_forward(const Int32Pair *backward_transitions,
                                    const DenominatorGraphTransition *transitions,
                                    int32_cuda num_sequences,
                                    int32_cuda num_hmm_states,
                                    const BaseFloat *probs,
                                    int32_cuda prob_stride,
                                    const BaseFloat *prev_alpha,
                                    BaseFloat *this_alpha) {
  // 'backward_transitions', indexed by hmm-state, consists of [start, end]
  // indexes into the 'transitions' array.  This gives us the info for
  // transitions *into* this state.  'probs' contains the exponentiated neural
  // net outputs; it has dimension num-output-indexes by num_sequences and its
  // stride is 'prob_stride'.  'prev_alpha' and 'this_alpha', which are
  // extracted from a larger matrix, both have dimension num-history-states by
  // num-sequences.

  // s is the index of the sequence within the minibatch,
  // from 0 .. num-egs-in-this-minibatch - 1.
  // h is the hmm-state index.
  int32_cuda s = threadIdx.x + blockIdx.x * blockDim.x,
      h  = blockIdx.y;
  if (s >= num_sequences)
    return;

  double this_tot_alpha = 0.0;
  const DenominatorGraphTransition
      *trans_iter = transitions + backward_transitions[h].first,
      *trans_end = transitions + backward_transitions[h].second;
  // Note: regarding this loop unrolling, I tried the automatic unrolling using
  // #pragma unroll 2 (after modifying the loop to have an integer index), but I
  // did not see any performance improvement, it was slightly slower.  So the
  // compiler must be doing something different than what I'm doing here.
  const int loop_unroll = 2;  // don't change this without changing the code
                              // below.
  for (; trans_iter + loop_unroll <= trans_end; trans_iter += loop_unroll) {
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
        prev_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat transition_prob1 = trans_iter[1].transition_prob;
    int32_cuda pdf_id1 = trans_iter[1].pdf_id,
        prev_hmm_state1 = trans_iter[1].hmm_state;
    BaseFloat pseudo_loglike0 = probs[pdf_id0 * prob_stride + s],
             this_prev_alpha0 = prev_alpha[prev_hmm_state0 * num_sequences + s],
              pseudo_loglike1 = probs[pdf_id1 * prob_stride + s],
             this_prev_alpha1 = prev_alpha[prev_hmm_state1 * num_sequences + s];

    this_tot_alpha += this_prev_alpha0 * transition_prob0 * pseudo_loglike0 +
                       this_prev_alpha1 * transition_prob1 * pseudo_loglike1;
  }
  if (trans_iter != trans_end) {
    // mop up the odd transition.
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
       prev_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat pseudo_loglike0 = probs[pdf_id0 * prob_stride + s],
             this_prev_alpha0 = prev_alpha[prev_hmm_state0 * num_sequences + s];
    this_tot_alpha += this_prev_alpha0 * transition_prob0 * pseudo_loglike0;
  }

  // Let arbitrary_scale be the inverse of the sum of all alpha values on-- the
  // previous frame this sum of all the alpha values is stored in the place that
  // we'd store the previous alpha for state-index equal to num_hmm_states
  // (i.e. one past the end).  We multiply this into all the
  // transition-probabilities from the previous frame to this frame, in both the
  // forward and backward passes, in order to keep the alphas in a good numeric
  // range.  This won't affect the posteriors, as it's just a constant factor
  // for each frame, but when computing the total likelihood we'll need to
  // compensate for it later on.
  BaseFloat arbitrary_scale =
      1.0 / prev_alpha[num_hmm_states * num_sequences + s];
  this_alpha[h * num_sequences + s] = this_tot_alpha * arbitrary_scale;
}


__global__
static void _cuda_chain_hmm_backward(const Int32Pair *forward_transitions,
                                     const DenominatorGraphTransition *transitions,
                                     int32_cuda num_sequences, int32_cuda num_hmm_states,
                                     const BaseFloat *probs, int32_cuda prob_stride,
                                     const BaseFloat *this_alpha, const BaseFloat *next_beta,
                                     BaseFloat *this_beta, BaseFloat *log_prob_deriv,
                                     int32_cuda log_prob_deriv_stride) {
  // 'forward_transitions', indexed by hmm-state, consists of [start, end]
  // indexes into the 'transition_info' array.  This is about the transitions
  // *out of* this state.  'probs' contains the exponentiated neural net
  // outputs; it has dimension num-output-indexes by num_sequences, and contains
  // just the observation probabilities for this time index.  Its stride is
  // prob_stride.
  // 'this_alpha', 'next_beta' and 'this_beta' all have dimension
  // num-history-states by num-sequences.
  // The beta probs are normalized in such a way (by multiplying by 1/(total-data-prob))
  // that to get occupation counts we don't need to multiply by 1/total-data-prob.
  // deriv_scale is a factor (e.g. -1.0 or -0.99) that we multiply these derivs by
  // while accumulating them.

  // s is the index of the sequence within the minibatch,
  // from 0 .. num-egs-in-this-minibatch - 1.
  // h is the hmm-state index.
  int32_cuda s = threadIdx.x + blockIdx.x * blockDim.x,
      h = blockIdx.y;
  if (s >= num_sequences)
    return;

  // See where arbitrary_scale is defined in the forward computation above, for
  // more explanation of inv_arbitrary_scale.
  BaseFloat this_alpha_prob = this_alpha[h * num_sequences + s],
      inv_arbitrary_scale =
      this_alpha[num_hmm_states * num_sequences + s];
  double tot_variable_factor = 0.0;

  BaseFloat occupation_factor = this_alpha_prob / inv_arbitrary_scale;
  const DenominatorGraphTransition
      *trans_iter = transitions + forward_transitions[h].first,
      *trans_end = transitions + forward_transitions[h].second;
  const int loop_unroll = 2;  // don't change this without changing the code
                              // below.
  for (; trans_iter + loop_unroll <= trans_end; trans_iter += loop_unroll) {
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
        next_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat transition_prob1 = trans_iter[1].transition_prob;
    int32_cuda pdf_id1 = trans_iter[1].pdf_id,
        next_hmm_state1 = trans_iter[1].hmm_state;
    BaseFloat variable_factor0 = transition_prob0 *
        next_beta[next_hmm_state0 * num_sequences + s] *
                    probs[pdf_id0 * prob_stride + s],
         variable_factor1 = transition_prob1 *
        next_beta[next_hmm_state1 * num_sequences + s] *
                    probs[pdf_id1 * prob_stride + s];
    tot_variable_factor += variable_factor0 + variable_factor1;
    BaseFloat occupation_prob0 = variable_factor0 * occupation_factor;
    atomic_add_thresholded(log_prob_deriv + (pdf_id0 * log_prob_deriv_stride + s),
                           occupation_prob0);
    BaseFloat occupation_prob1 = variable_factor1 * occupation_factor;
    atomic_add_thresholded(log_prob_deriv + (pdf_id1 * log_prob_deriv_stride + s),
                           occupation_prob1);
  }
  if (trans_iter != trans_end) {
    // mop up the odd transition.
    BaseFloat transition_prob0 = trans_iter[0].transition_prob;
    int32_cuda pdf_id0 = trans_iter[0].pdf_id,
        next_hmm_state0 = trans_iter[0].hmm_state;
    BaseFloat variable_factor0 = transition_prob0 *
        next_beta[next_hmm_state0 * num_sequences + s] *
                      probs[pdf_id0 * prob_stride + s];
    tot_variable_factor += variable_factor0;
    BaseFloat occupation_prob0 = variable_factor0 * occupation_factor;
    atomic_add_thresholded(log_prob_deriv + (pdf_id0 * log_prob_deriv_stride + s),
                           occupation_prob0);
  }
  BaseFloat beta = tot_variable_factor / inv_arbitrary_scale;
  this_beta[h * num_sequences + s] = beta;
}


void cuda_chain_hmm_forward(dim3 Gr, dim3 Bl,
                            const Int32Pair *backward_transitions,
                            const DenominatorGraphTransition *transitions,
                            int32_cuda num_sequences,
                            int32_cuda num_hmm_states,
                            const BaseFloat *probs, int32_cuda prob_stride,
                            const BaseFloat *prev_alpha,
                            BaseFloat *this_alpha) {
  _cuda_chain_hmm_forward<<<Gr,Bl>>>(backward_transitions, transitions,
                                     num_sequences, num_hmm_states,
                                     probs, prob_stride,
                                     prev_alpha, this_alpha);
}

void cuda_chain_hmm_backward(dim3 Gr, dim3 Bl,
                             const Int32Pair *forward_transitions,
                             const DenominatorGraphTransition *transitions,
                             int32_cuda num_sequences,
                             int32_cuda num_hmm_states,
                             const BaseFloat *probs, int32_cuda prob_stride,
                             const BaseFloat *this_alpha, const BaseFloat *next_beta,
                             BaseFloat *this_beta,
                             BaseFloat *log_prob_deriv,
                             int32_cuda log_prob_deriv_stride) {
  _cuda_chain_hmm_backward<<<Gr,Bl>>>(forward_transitions, transitions,
                                      num_sequences, num_hmm_states,
                                      probs, prob_stride,
                                      this_alpha, next_beta,
                                      this_beta, log_prob_deriv,
                                      log_prob_deriv_stride);
}


// See documentation for PenalizeOutOfRange() in chain-training.cc to see what
// this is about.
__global__
static void _penalize_out_of_range(
    BaseFloat limit, BaseFloat scale, const BaseFloat *in_data, MatrixDim dim,
    int out_stride, BaseFloat *out_deriv) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int in_index = i + j * dim.stride,
      out_index = i + j * out_stride;
  if (i < dim.cols && j < dim.rows) {
    BaseFloat val = in_data[in_index];
    if (val < -limit) {
      out_deriv[out_index] -= scale * (val + limit);
    } else if (val > limit) {
      out_deriv[out_index] -= scale * (val - limit);
    }
  }
}

void cuda_penalize_out_of_range(dim3 Gr, dim3 Bl, BaseFloat limit,
                                BaseFloat scale, const BaseFloat *in_data,
                                MatrixDim dim, int out_stride,
                                BaseFloat *out_deriv) {
  _penalize_out_of_range<<<Gr,Bl>>>(limit, scale, in_data,
                                    dim, out_stride, out_deriv);
}
